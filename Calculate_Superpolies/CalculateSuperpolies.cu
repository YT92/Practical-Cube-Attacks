#include "hip/hip_runtime.h"
//�����ǽṹ��,u32,֮��ֻ��Ҫ�������Ĺ����ͺ���
#include<Windows.h>
#include "trivium.h"
#include "ecrypt-sync.h"
#include "hip/hip_runtime.h"
#include ""
//#include"cube attack.h"
#include <stdio.h>
#include<time.h>
#include<math.h>
#include"randomkey.h"
#define imin(a,b) (a<b?a:b)
#define TYPE unsigned int
//__constant__ u32 iv[10];
const int threadsPerBlock=1024;
//const int threadsPerBlock=32;
u8 firstoutput=1;
//__constant__ ECRYPT_ctx ctx;
//ѡ�������Կ,��host�˺�������,Ȼ�����ɵ������Կ���뵽
void choose_random_key(u32 KEY[])
{
	u8 i=0;
	u8 j=0;

	KEY[0]=rand()&0x000000FF;
 for(i=1;i<10;i++)
   {
	KEY[i]=rand()&0x000000FF;
LOOP: for (j=0;j<i;j++)
	  {
		if (KEY[i]==KEY[j]) 
		{
         KEY[i]=rand()&0x000000FF;
		 goto LOOP;
		}
	  }
   }
}

//void Moebius(TYPE *Tab, TYPE size)
void Moebius(TYPE *Tab, unsigned __int64 size)
{
	unsigned __int64  Wsize;
	//TYPE i,i0,i1; 
	//TYPE step;
	TYPE temp;
	unsigned __int64 i,i0,i1;
	unsigned __int64 step;
	Wsize=size/(8*sizeof(TYPE));
	
	/*Moebius transform for high order bits, using word ops*/
	for (step=1;step<Wsize;step<<=1) {
		for (i1=0;i1<Wsize;i1+=2*step) {
			for (i0=0;i0<step;i0++) {
				i=i1+i0;
				Tab[i+step]^=Tab[i];
			}
		}
	}
	
/*Moebius transform for low order bits, within words*/
/* Assumes 8*sizeof(TYPE)=32 */
	for(i=0;i<Wsize;i++) {
		TYPE tmp;
		tmp=Tab[i];
		tmp^=(tmp<<16);
		tmp^=(tmp&0xff00ff)<<8;
		tmp^=(tmp&0xf0f0f0f)<<4;
		tmp^=(tmp&0x33333333)<<2;
		tmp^=(tmp&0x55555555)<<1;
		Tab[i]=tmp;
	}
}
__device__ u32 reverse_word(u32 a)
{
	u32 b = 0;

	b = ((a&0x000000FF)<<24)^((a&0x0000FF00)<<8)^((a&0x00FF0000)>>8)^((a&0xFF000000)>>24);
	b = ((b&0x01010101)<<7)^((b&0x02020202)<<5)^((b&0x04040404)<<3)^((b&0x08080808)<<1)^((b&0x10101010)>>1)^((b&0x20202020)>>3)^((b&0x40404040)>>5)^((b&0x80808080)>>7);

	return(b);
}

 __device__ u32 Trivium_update_funcion_word(ECRYPT_ctx* ctx)
{
	u32 t1,t2,t3;
	u32 s66,s93,s162,s177,s243,s288,s91,s92,s171,s175,s176,s264,s286,s287,s69;
	u32 z;

	s66 = (ctx->s[2]<<30)|(ctx->s[1]>>2);//���֮���൱��ֱ�Ӽ���
	s93 = (ctx->s[2]<<3)|(ctx->s[1]>>29);
	s162 = (ctx->s[5]<<27)|(ctx->s[4]>>5);
	s177 = (ctx->s[5]<<12)|(ctx->s[4]>>20);
	s243 = (ctx->s[8]<<30)|(ctx->s[7]>>2);
	s288 = (ctx->s[9]<<17)| (ctx->s[8]>>15);

	t1 = s66^s93;

	t2 = s162^s177;

	t3 = s243^s288;

	z = t1^t2^t3;
	
	//����t1,t2,t3
	//t1 = t1 + s91s92 + s171
	s91 = (ctx->s[2]<<5)|(ctx->s[1]>>27);
	s92 = (ctx->s[2]<<4)|(ctx->s[1]>>28);
	s171 = (ctx->s[5]<<18)|(ctx->s[4]>>14);

	t1 ^= (s91&s92)^s171;

	//t2 = t2 + s175s176 + s264
	s175 = (ctx->s[5]<<14)|(ctx->s[4]>>18);
	s176 = (ctx->s[5]<<13)|(ctx->s[4]>>19);
	s264 = (ctx->s[8]<<9)|(ctx->s[7]>>23);

	t2 ^= (s175&s176)^s264;

	//t3 = t3 + s286s287 + s69
	s286 = (ctx->s[9]<<19)|(ctx->s[8]>>13);
	s287 = (ctx->s[9]<<18)|(ctx->s[8]>>14);
	s69 = (ctx->s[2]<<27)|(ctx->s[1]>>5);

	t3 ^= (s286&s287)^s69;

	// update register 1
	ctx->s[2] = (ctx->s[1])&(0x1FFFFFFF);
	ctx->s[1] = ctx->s[0];
	ctx->s[0] = t3;

	//update register 2
	ctx->s[5] = ctx->s[4]&(0x000FFFFF);
	ctx->s[4] = ctx->s[3];
	ctx->s[3] = t1;

	//update register 3
	ctx->s[9] = ctx->s[8]&(0x00007FFF);
	ctx->s[8] = ctx->s[7];
	ctx->s[7] = ctx->s[6];
	ctx->s[6] = t2;
	//printf("%d  ",z);

	
	return z;
}

 __device__ u8 Trivium_update_funcion_bit(ECRYPT_ctx* ctx)
{
	u32 t1,t2,t3;
	u32 s66,s93,s162,s177,s243,s288,s91,s92,s171,s175,s176,s264,s286,s287,s69;
	u32 z;

	s66 = ((ctx->s[2]<<30)&0x80000000)>>31;//��λ�Ժ�ֻȡ���λ
	s93 = ((ctx->s[2]<<3)&0x80000000)>>31;
	s162 = ((ctx->s[5]<<27)&0x80000000)>>31;
	s177 = ((ctx->s[5]<<12)&0x80000000)>>31;
	s243 = ((ctx->s[8]<<30)&0x80000000)>>31;
	s288 = ((ctx->s[9]<<17)&0x80000000)>>31;

	//s(66)+s(93)
	t1 = s66^s93;

	//s(162)+s(177)
	t2 = s162^s177;

	//s(243)+s(288)
	t3 = s243^s288;

	z = t1^t2^t3;
	//����t1,t2,t3
	//t1 = t1 + s91s92 + s171
	s91 = ((ctx->s[2]<<5)&0x80000000)>>31;
	s92 = ((ctx->s[2]<<4)&0x80000000)>>31;
	s171 = ((ctx->s[5]<<18)&0x80000000)>>31;

	t1 ^= (s91&s92)^s171;

	//t2 = t2 + s175s176 + s264
	s175 = ((ctx->s[5]<<14)&0x80000000)>>31;
	s176 = ((ctx->s[5]<<13)&0x80000000)>>31;
	s264 = ((ctx->s[8]<<9)&0x80000000)>>31;

	t2 ^= (s175&s176)^s264;

	//t3 = t3 + s286s287 + s69
	s286 = ((ctx->s[9]<<19)&0x80000000)>>31;
	s287 = ((ctx->s[9]<<18)&0x80000000)>>31;
	s69 = ((ctx->s[2]<<27)&0x80000000)>>31;
	t3 ^= (s286&s287)^s69;
	// update register 1
	//S[0]������һλ,Ȼ�������²�����һ������//((ctx->s[1]&0x80000000)>>31)
	ctx->s[2]=((ctx->s[2]<<1)|(((ctx->s[1]&0x80000000)>>31)))&(0x1FFFFFFF);
	ctx->s[1]=(ctx->s[1]<<1)|(((ctx->s[0]&0x80000000)>>31));
	ctx->s[0]=(ctx->s[0]<<1)|t3;
	//update register 2
	ctx->s[5]=((ctx->s[5]<<1)|(((ctx->s[4]&0x80000000)>>31)))&(0x000FFFFF);
	ctx->s[4]=(ctx->s[4]<<1)|(((ctx->s[3]&0x80000000)>>31));
	ctx->s[3]=(ctx->s[3]<<1)|t1;
	//update register 3
	ctx->s[9]=((ctx->s[9]<<1)|(((ctx->s[8]&0x80000000)>>31)))&(0x00007FFF);
	ctx->s[8]=(ctx->s[8]<<1)|(((ctx->s[7]&0x80000000)>>31));
	ctx->s[7]=(ctx->s[7]<<1)|(((ctx->s[6]&0x80000000)>>31));
	ctx->s[6]=(ctx->s[6]<<1)|t2;
	return z;
}
  __device__ void ECRYPT_ivsetup_bit( ECRYPT_ctx* ctx,  u32* iv, u32 roundnum)
  {
  u8 i;
  u32 roundnum_word;
  u32 roundnum_bit;
 
  ctx->s[0] = ctx->key[0]^(ctx->key[1]<<8)^(ctx->key[2]<<16)^(ctx->key[3]<<24);
  ctx->s[1] = ctx->key[4]^(ctx->key[5]<<8)^(ctx->key[6]<<16)^(ctx->key[7]<<24);
  ctx->s[2] = ctx->key[8]^(ctx->key[9]<<8);

  ctx->s[3] = iv[0]^(iv[1]<<8)^(iv[2]<<16)^(iv[3]<<24);
  ctx->s[4] = iv[4]^(iv[5]<<8)^(iv[6]<<16)^(iv[7]<<24);
  ctx->s[5] = iv[8]^(iv[9]<<8);

  ctx->s[6] = 0;
  ctx->s[7] = 0;
  ctx->s[8] = 0;
  ctx->s[9] = 0x00007000;
  
  //32*36 = 1152
  roundnum_word=roundnum/32;
  roundnum_bit=roundnum%32;
  for(i=0;i<roundnum_word;i++)
	Trivium_update_funcion_word( ctx);
  for(i=0;i<roundnum_bit;i++)
	Trivium_update_funcion_bit(ctx);
}
 __device__ void ECRYPT_keysetup( ECRYPT_ctx* ctx,   u32* key,  u32 keysize, u32 ivsize)
{
  u8 i;

  ctx->keylen = 10;
  ctx->ivlen = 10;

  for (i = 0; i < ctx->keylen; ++i)
    ctx->key[i] = key[i];
}

 __device__ void ECRYPT_keystream_words( ECRYPT_ctx* ctx,  u32* keystream,  u32 length)               
{
	u32 j;
	u32 z;

	for(j=0;j<length;j++)
	{
		z = Trivium_update_funcion_word(ctx);
		keystream[j] = reverse_word(z);
	}
}

//  __device__ void ECRYPT_keystream_wordsV2(u32* keystream,u32* iv, u32 *key, u32 roundnum)               
//{
//	u32 z1,j;
//	//z = Trivium_update_funcion_word(ctx);
//	//�����ú���ֱ��д
//  u32 roundnum_word;
//  u32 roundnum_bit;
//  u32 t1,t2,t3,i;
//  u32 s[10];
//  u32 s66,s93,s162,s177,s243,s288,s91,s92,s171,s175,s176,s264,s286,s287,s69;
//  u32 z;
// s[0] = key[0]^(key[1]<<8)^(key[2]<<16)^(key[3]<<24);
// s[1] = key[4]^(key[5]<<8)^(key[6]<<16)^(key[7]<<24);
// s[2] = key[8]^(key[9]<<8);
//
// s[3] = iv[0]^(iv[1]<<8)^(iv[2]<<16)^(iv[3]<<24);
// s[4] = iv[4]^(iv[5]<<8)^(iv[6]<<16)^(iv[7]<<24);
// s[5] = iv[8]^(iv[9]<<8);
//
// s[6] = 0;
// s[7] = 0;
// s[8] = 0;
// s[9] = 0x00007000;
//  
// // 32*36 = 1152
//  roundnum_word=roundnum/32;
//  roundnum_bit=roundnum%32;
//  for(i=0;i<roundnum_word;i++)
//  {
//
//	s66 = (s[2]<<30)|(s[1]>>2);//���֮���൱��ֱ�Ӽ���
//	s93 = (s[2]<<3)|(s[1]>>29);
//	s162 = (s[5]<<27)|(s[4]>>5);
//	s177 = (s[5]<<12)|(s[4]>>20);
//	s243 = (s[8]<<30)|(s[7]>>2);
//	s288 = (s[9]<<17)| (s[8]>>15);
//
//	t1 = s66^s93;
//
//	t2 = s162^s177;
//
//	t3 = s243^s288;
//
//	z = t1^t2^t3;
//	
//	//����t1,t2,t3
////	t1 = t1 + s91s92 + s171
//	s91 = (s[2]<<5)|(s[1]>>27);
//	s92 = (s[2]<<4)|(s[1]>>28);
//	s171 = (s[5]<<18)|(s[4]>>14);
//
//	t1 ^= (s91&s92)^s171;
//
////	t2 = t2 + s175s176 + s264
//	s175 = (s[5]<<14)|(s[4]>>18);
//	s176 = (s[5]<<13)|(s[4]>>19);
//	s264 = (s[8]<<9)|(s[7]>>23);
//
//	t2 ^= (s175&s176)^s264;
//
////	t3 = t3 + s286s287 + s69
//	s286 = (s[9]<<19)|(s[8]>>13);
//	s287 = (s[9]<<18)|(s[8]>>14);
//	s69 = (s[2]<<27)|(s[1]>>5);
//
//	t3 ^= (s286&s287)^s69;
//
//	// update register 1
//	s[2] = (s[1])&(0x1FFFFFFF);
//	s[1] =s[0];
//	s[0] = t3;
//
////	update register 2
//	s[5] =s[4]&(0x000FFFFF);
//	s[4] =s[3];
//	s[3] = t1;
//
////	update register 3
//	s[9] =s[8]&(0x00007FFF);
//	s[8] =s[7];
//	s[7] =s[6];
//	s[6] = t2;
//  }
//  if(roundnum_bit!=0)
//  {
//	s66 = (s[2]<<30)|(s[1]>>2);//���֮���൱��ֱ�Ӽ���
//	s93 = (s[2]<<3)|(s[1]>>29);
//	s162 = (s[5]<<27)|(s[4]>>5);
//	s177 = (s[5]<<12)|(s[4]>>20);
//	s243 = (s[8]<<30)|(s[7]>>2);
//	s288 = (s[9]<<17)| (s[8]>>15);
//
//	t1 = s66^s93;
//
//	t2 = s162^s177;
//
//	t3 = s243^s288;
//
//	z = t1^t2^t3;
//	z1=z;
//	//����t1,t2,t3
//	//t1 = t1 + s91s92 + s171
//	s91 = (s[2]<<5)|(s[1]>>27);
//	s92 = (s[2]<<4)|(s[1]>>28);
//	s171 = (s[5]<<18)|(s[4]>>14);
//
//	t1 ^= (s91&s92)^s171;
//
//	//t2 = t2 + s175s176 + s264
//	s175 = (s[5]<<14)|(s[4]>>18);
//	s176 = (s[5]<<13)|(s[4]>>19);
//	s264 = (s[8]<<9)|(s[7]>>23);
//
//	t2 ^= (s175&s176)^s264;
//
//	//t3 = t3 + s286s287 + s69
//	s286 = (s[9]<<19)|(s[8]>>13);
//	s287 = (s[9]<<18)|(s[8]>>14);
//	s69 = (s[2]<<27)|(s[1]>>5);
//
//	t3 ^= (s286&s287)^s69;
//
//	// update register 1
//	s[2] = (s[1])&(0x1FFFFFFF);
//	s[1] =s[0];
//	s[0] = t3;
//
//	//update register 2
//	s[5] =s[4]&(0x000FFFFF);
//	s[4] =s[3];
//	s[3] = t1;
//
//	//update register 3
//	s[9] =s[8]&(0x00007FFF);
//	s[8] =s[7];
//	s[7] =s[6];
//	s[6] = t2;
//  }
//	s66 = (s[2]<<30)|(s[1]>>2);//���֮���൱��ֱ�Ӽ���
//	s93 = (s[2]<<3)|(s[1]>>29);
//	s162 = (s[5]<<27)|(s[4]>>5);
//	s177 = (s[5]<<12)|(s[4]>>20);
//	s243 = (s[8]<<30)|(s[7]>>2);
//	s288 = (s[9]<<17)| (s[8]>>15);
//	t1 = s66^s93;
//	t2 = s162^s177;
//	t3 = s243^s288;
//	z = t1^t2^t3;
//	if(roundnum_bit!=0)
//		z=(z1<<roundnum_bit)|(z>>(32-roundnum_bit));
//	keystream[0]=reverse_word(z);
//}



 __device__ void ECRYPT_keystream_wordsV2(u32* keystream,u32* iv, u32 *key, u32 roundnum)               
{
	u32 z1;
	//z = Trivium_update_funcion_word(ctx);
	//�����ú���ֱ��д
  u32 roundnum_word;
  u32 roundnum_bit;
  u32 t1,t2,t3,i;
  u32 s[10];
  //u32 temp1,temp2,temp3,temp4,temp5,temp6,s91,s92,s171,s175,s176,s264,s286,s287,s69;
  u32 temp1,temp2,temp3,temp4,temp5,temp6;
  u32 z;
 s[0] = key[0]^(key[1]<<8)^(key[2]<<16)^(key[3]<<24);
 s[1] = key[4]^(key[5]<<8)^(key[6]<<16)^(key[7]<<24);
 s[2] = key[8]^(key[9]<<8);

 s[3] = iv[0]^(iv[1]<<8)^(iv[2]<<16)^(iv[3]<<24);
 s[4] = iv[4]^(iv[5]<<8)^(iv[6]<<16)^(iv[7]<<24);
 s[5] = iv[8]^(iv[9]<<8);

 s[6] = 0;
 s[7] = 0;
 s[8] = 0;
 s[9] = 0x00007000;
  
 // 32*36 = 1152
  roundnum_word=roundnum/32;
  roundnum_bit=roundnum%32;
  for(i=0;i<roundnum_word;i++)
  {

	temp1 = (s[2]<<30)|(s[1]>>2);//���֮���൱��ֱ�Ӽ���
	temp2 = (s[2]<<3)|(s[1]>>29);
	temp3 = (s[5]<<27)|(s[4]>>5);
	temp4 = (s[5]<<12)|(s[4]>>20);
	temp5 = (s[8]<<30)|(s[7]>>2);
	temp6 = (s[9]<<17)| (s[8]>>15);

	t1 = temp1^temp2;

	t2 = temp3^temp4;

	t3 = temp5^temp6;

	//z = t1^t2^t3;
	
	//����t1,t2,t3
//	t1 = t1 + s91s92 + s171
	temp1 = (s[2]<<5)|(s[1]>>27);
	temp2 = (s[2]<<4)|(s[1]>>28);
	temp3 = (s[5]<<18)|(s[4]>>14);

	t1 ^= (temp1&temp2)^temp3;

//	t2 = t2 + s175s176 + s264
	temp1 = (s[5]<<14)|(s[4]>>18);
	temp2 = (s[5]<<13)|(s[4]>>19);
	temp3 = (s[8]<<9)|(s[7]>>23);

	t2 ^= (temp1&temp2)^temp3;

//	t3 = t3 + s286s287 + s69
	temp1 = (s[9]<<19)|(s[8]>>13);
	temp2 = (s[9]<<18)|(s[8]>>14);
	temp3 = (s[2]<<27)|(s[1]>>5);

	t3 ^= (temp1&temp2)^temp3;

	// update register 1
	s[2] = (s[1])&(0x1FFFFFFF);
	s[1] =s[0];
	s[0] = t3;

//	update register 2
	s[5] =s[4]&(0x000FFFFF);
	s[4] =s[3];
	s[3] = t1;

//	update register 3
	s[9] =s[8]&(0x00007FFF);
	s[8] =s[7];
	s[7] =s[6];
	s[6] = t2;
  }
  if(roundnum_bit!=0)
  {
	temp1 = (s[2]<<30)|(s[1]>>2);//���֮���൱��ֱ�Ӽ���
	temp2 = (s[2]<<3)|(s[1]>>29);
	temp3 = (s[5]<<27)|(s[4]>>5);
	temp4 = (s[5]<<12)|(s[4]>>20);
	temp5 = (s[8]<<30)|(s[7]>>2);
	temp6 = (s[9]<<17)| (s[8]>>15);

	t1= temp1^temp2;
	t2= temp3^temp4;
	t3= temp5^temp6;
	z1 = temp1^temp2^temp3^temp4^temp5^temp6;
	//z1=z;
	//����t1,t2,t3
	//t1 = t1 + s91s92 + s171
	temp1 = (s[2]<<5)|(s[1]>>27);
	temp2 = (s[2]<<4)|(s[1]>>28);
	temp3 = (s[5]<<18)|(s[4]>>14);

	t1 ^= (temp1&temp2)^temp3;

	//t2 = t2 + s175s176 + s264
	temp1 = (s[5]<<14)|(s[4]>>18);
	temp2 = (s[5]<<13)|(s[4]>>19);
	temp3 = (s[8]<<9)|(s[7]>>23);

	t2 ^= (temp1&temp2)^temp3;

	//t3 = t3 + s286s287 + s69
	temp1 = (s[9]<<19)|(s[8]>>13);
	temp2 = (s[9]<<18)|(s[8]>>14);
	temp3 = (s[2]<<27)|(s[1]>>5);

	t3 ^= (temp1&temp2)^temp3;

	// update register 1
	s[2] = (s[1])&(0x1FFFFFFF);
	s[1] =s[0];
	s[0] = t3;

	//update register 2
	s[5] =s[4]&(0x000FFFFF);
	s[4] =s[3];
	s[3] = t1;

	//update register 3
	s[9] =s[8]&(0x00007FFF);
	s[8] =s[7];
	s[7] =s[6];
	s[6] = t2;
  }
	temp1 = (s[2]<<30)|(s[1]>>2);//���֮���൱��ֱ�Ӽ���
	temp2 = (s[2]<<3)|(s[1]>>29);
	temp3 = (s[5]<<27)|(s[4]>>5);
	temp4 = (s[5]<<12)|(s[4]>>20);
	temp5 = (s[8]<<30)|(s[7]>>2);
	temp6 = (s[9]<<17)| (s[8]>>15);
	z = temp1^temp2^temp3^temp4^temp5^temp6;
	if(roundnum_bit!=0)
		z=(z1<<roundnum_bit)|(z>>(32-roundnum_bit));
	//keystream[0]=reverse_word(z);
	temp1=0;
	temp1 = ((z&0x000000FF)<<24)^((z&0x0000FF00)<<8)^((z&0x00FF0000)>>8)^((z&0xFF000000)>>24);
	temp1 = ((temp1&0x01010101)<<7)^((temp1&0x02020202)<<5)^((temp1&0x04040404)<<3)^((temp1&0x08080808)<<1)^((temp1&0x10101010)>>1)^((temp1&0x20202020)>>3)^((temp1&0x40404040)>>5)^((temp1&0x80808080)>>7);
	keystream[0]=temp1;
}

 __device__ void ECRYPT_keystream_wordsV3(u32* keystream,u32* iv, u32 *key, u32 roundnum)               
{
	u32 z1;
	//z = Trivium_update_funcion_word(ctx);
	//�����ú���ֱ��д
  u32 roundnum_word;
  u32 roundnum_bit;
  u32 t1,t2,t3,i;
  u32 s0,s1,s2,s3,s4,s5,s6,s7,s8,s9;
  //u32 temp1,temp2,temp3,temp4,temp5,temp6,s91,s92,s171,s175,s176,s264,s286,s287,s69;
  u32 temp1,temp2,temp3,temp4,temp5,temp6;
  u32 z;
 s0 = key[0]^(key[1]<<8)^(key[2]<<16)^(key[3]<<24);
 s1 = key[4]^(key[5]<<8)^(key[6]<<16)^(key[7]<<24);
 s2 = key[8]^(key[9]<<8);

 s3 = iv[0]^(iv[1]<<8)^(iv[2]<<16)^(iv[3]<<24);
 s4 = iv[4]^(iv[5]<<8)^(iv[6]<<16)^(iv[7]<<24);
 s5 = iv[8]^(iv[9]<<8);

 s6 = 0;
 s7 = 0;
 s8 = 0;
 s9 = 0x00007000;
  
 // 32*36 = 1152
  roundnum_word=roundnum/32;
  roundnum_bit=roundnum%32;
  for(i=0;i<roundnum_word;i++)
  {

	temp1 = (s2<<30)|(s1>>2);//���֮���൱��ֱ�Ӽ���
	temp2 = (s2<<3)|(s1>>29);
	temp3 = (s5<<27)|(s4>>5);
	temp4 = (s5<<12)|(s4>>20);
	temp5 = (s8<<30)|(s7>>2);
	temp6 = (s9<<17)| (s8>>15);

	t1 = temp1^temp2;//((s2<<30)|(s1>>2))^((s2<<3)|(s1>>29))^(((s2<<5)|(s1>>27))&((s2<<4)|(s1>>28)))^(s5<<18)|(s4>>14);

	t2 = temp3^temp4;//((s5<<27)|(s4>>5))^((s5<<12)|(s4>>20))^((s5<<14)|(s4>>18))&((s5<<13)|(s4>>19))^((s8<<9)|(s7>>23));

	t3 = temp5^temp6;//((s8<<30)|(s7>>2))^((s9<<17)| (s8>>15))^

	//z = t1^t2^t3;
	
	//����t1,t2,t3
//	t1 = t1 + s91s92 + s171
	temp1 = (s2<<5)|(s1>>27);//(((s2<<5)|(s1>>27))&((s2<<4)|(s1>>28)))^(s5<<18)|(s4>>14);
	temp2 = (s2<<4)|(s1>>28);
	temp3 = (s5<<18)|(s4>>14);

	t1 ^= (temp1&temp2)^temp3;

//	t2 = t2 + s175s176 + s264
	temp1 = (s5<<14)|(s4>>18);//(((s5<<14)|(s4>>18))&((s5<<13)|(s4>>19)))^((s8<<9)|(s7>>23));
	temp2 = (s5<<13)|(s4>>19);
	temp3 = (s8<<9)|(s7>>23);

	t2 ^= (temp1&temp2)^temp3;

//	t3 = t3 + s286s287 + s69
	temp1 = (s9<<19)|(s8>>13);//(()&())^()
	temp2 = (s9<<18)|(s8>>14);
	temp3 = (s2<<27)|(s1>>5);

	t3 ^= (temp1&temp2)^temp3;

	// update register 1
	s2 = (s1)&(0x1FFFFFFF);
	s1 =s0;
	s0 = t3;

//	update register 2
	s5 =s4&(0x000FFFFF);
	s4 =s3;
	s3 = t1;

//	update register 3
	s9 =s8&(0x00007FFF);
	s8 =s7;
	s7 =s6;
	s6 = t2;
  }
  if(roundnum_bit!=0)
  {
	temp1 = (s2<<30)|(s1>>2);//���֮���൱��ֱ�Ӽ���
	temp2 = (s2<<3)|(s1>>29);
	temp3 = (s5<<27)|(s4>>5);
	temp4 = (s5<<12)|(s4>>20);
	temp5 = (s8<<30)|(s7>>2);
	temp6 = (s9<<17)| (s8>>15);

	t1= temp1^temp2;
	t2= temp3^temp4;
	t3= temp5^temp6;
	z1 = temp1^temp2^temp3^temp4^temp5^temp6;
	//z1=z;
	//����t1,t2,t3
	//t1 = t1 + s91s92 + s171
	temp1 = (s2<<5)|(s1>>27);
	temp2 = (s2<<4)|(s1>>28);
	temp3 = (s5<<18)|(s4>>14);

	t1 ^= (temp1&temp2)^temp3;

	//t2 = t2 + s175s176 + s264
	temp1 = (s5<<14)|(s4>>18);
	temp2 = (s5<<13)|(s4>>19);
	temp3 = (s8<<9)|(s7>>23);

	t2 ^= (temp1&temp2)^temp3;

	//t3 = t3 + s286s287 + s69
	temp1 = (s9<<19)|(s8>>13);
	temp2 = (s9<<18)|(s8>>14);
	temp3 = (s2<<27)|(s1>>5);

	t3 ^= (temp1&temp2)^temp3;

	// update register 1
	s2 = (s1)&(0x1FFFFFFF);
	s1 =s0;
	s0 = t3;

	//update register 2
	s5 =s4&(0x000FFFFF);
	s4 =s3;
	s3 = t1;

	//update register 3
	s9 =s8&(0x00007FFF);
	s8 =s7;
	s7 =s6;
	s6 = t2;
  }
	temp1 = (s2<<30)|(s1>>2);//���֮���൱��ֱ�Ӽ���
	temp2 = (s2<<3)|(s1>>29);
	temp3 = (s5<<27)|(s4>>5);
	temp4 = (s5<<12)|(s4>>20);
	temp5 = (s8<<30)|(s7>>2);
	temp6 = (s9<<17)| (s8>>15);
	z = temp1^temp2^temp3^temp4^temp5^temp6;
	if(roundnum_bit!=0)
		z=(z1<<roundnum_bit)|(z>>(32-roundnum_bit));
	//keystream[0]=reverse_word(z);
	temp1=0;
	temp1 = ((z&0x000000FF)<<24)^((z&0x0000FF00)<<8)^((z&0x00FF0000)>>8)^((z&0xFF000000)>>24);
	temp1 = ((temp1&0x01010101)<<7)^((temp1&0x02020202)<<5)^((temp1&0x04040404)<<3)^((temp1&0x08080808)<<1)^((temp1&0x10101010)>>1)^((temp1&0x20202020)>>3)^((temp1&0x40404040)>>5)^((temp1&0x80808080)>>7);
	keystream[0]=temp1;
}

 __device__ void ECRYPT_keystream_wordsV4(u32* keystream, u32 *key, u32 roundnum,u32 cube[], u32 dim, u32 k, u32 loc)               
{
	u32 z1;
	//z = Trivium_update_funcion_word(ctx);
	//�����ú���ֱ��д
  u32 roundnum_word;
  u32 roundnum_bit;
  u32 t1,t2,t3,i,j,l;
  u32 s0,s1,s2,s3,s4,s5,s6,s7,s8,s9;
  //u32 temp1,temp2,temp3,temp4,temp5,temp6,s91,s92,s171,s175,s176,s264,s286,s287,s69;
  u32 temp1,temp2,temp3,temp4,temp5,temp6;
  u32 z;
  u32 iv[10];
  for(i=0;i<10;i++)
	  iv[i]=0;
  i=k;
  for (j=0;j<dim;j++) 
			iv[cube[j]>>3] |= ( ( (i>>j) & 0x00000001 ) << (cube[j] & 0x07) );
	 j=loc;
	for(l=0;l<6;l++)
	{
		i=dim+l;
		iv[cube[i]>>3]|=(j&0x01)<<(cube[i]&0x07);
		j>>=1;
	}
 s0 = key[0]^(key[1]<<8)^(key[2]<<16)^(key[3]<<24);
 s1 = key[4]^(key[5]<<8)^(key[6]<<16)^(key[7]<<24);
 s2 = key[8]^(key[9]<<8);

 s3 = iv[0]^(iv[1]<<8)^(iv[2]<<16)^(iv[3]<<24);
 s4 = iv[4]^(iv[5]<<8)^(iv[6]<<16)^(iv[7]<<24);
 s5 = iv[8]^(iv[9]<<8);

 s6 = 0;
 s7 = 0;
 s8 = 0;
 s9 = 0x00007000;
  
 // 32*36 = 1152
  roundnum_word=roundnum/32;
  roundnum_bit=roundnum%32;
  for(i=0;i<roundnum_word;i++)
  {

	temp1 = (s2<<30)|(s1>>2);//���֮���൱��ֱ�Ӽ���
	temp2 = (s2<<3)|(s1>>29);
	temp3 = (s5<<27)|(s4>>5);
	temp4 = (s5<<12)|(s4>>20);
	temp5 = (s8<<30)|(s7>>2);
	temp6 = (s9<<17)| (s8>>15);

	t1 = temp1^temp2;//((s2<<30)|(s1>>2))^((s2<<3)|(s1>>29))^(((s2<<5)|(s1>>27))&((s2<<4)|(s1>>28)))^(s5<<18)|(s4>>14);

	t2 = temp3^temp4;//((s5<<27)|(s4>>5))^((s5<<12)|(s4>>20))^((s5<<14)|(s4>>18))&((s5<<13)|(s4>>19))^((s8<<9)|(s7>>23));

	t3 = temp5^temp6;//((s8<<30)|(s7>>2))^((s9<<17)| (s8>>15))^

	//z = t1^t2^t3;
	
	//����t1,t2,t3
//	t1 = t1 + s91s92 + s171
	temp1 = (s2<<5)|(s1>>27);//(((s2<<5)|(s1>>27))&((s2<<4)|(s1>>28)))^(s5<<18)|(s4>>14);
	temp2 = (s2<<4)|(s1>>28);
	temp3 = (s5<<18)|(s4>>14);

	t1 ^= (temp1&temp2)^temp3;

//	t2 = t2 + s175s176 + s264
	temp1 = (s5<<14)|(s4>>18);//(((s5<<14)|(s4>>18))&((s5<<13)|(s4>>19)))^((s8<<9)|(s7>>23));
	temp2 = (s5<<13)|(s4>>19);
	temp3 = (s8<<9)|(s7>>23);

	t2 ^= (temp1&temp2)^temp3;

//	t3 = t3 + s286s287 + s69
	temp1 = (s9<<19)|(s8>>13);//(()&())^()
	temp2 = (s9<<18)|(s8>>14);
	temp3 = (s2<<27)|(s1>>5);

	t3 ^= (temp1&temp2)^temp3;

	// update register 1
	s2 = (s1)&(0x1FFFFFFF);
	s1 =s0;
	s0 = t3;

//	update register 2
	s5 =s4&(0x000FFFFF);
	s4 =s3;
	s3 = t1;

//	update register 3
	s9 =s8&(0x00007FFF);
	s8 =s7;
	s7 =s6;
	s6 = t2;
  }
  if(roundnum_bit!=0)
  {
	temp1 = (s2<<30)|(s1>>2);//���֮���൱��ֱ�Ӽ���
	temp2 = (s2<<3)|(s1>>29);
	temp3 = (s5<<27)|(s4>>5);
	temp4 = (s5<<12)|(s4>>20);
	temp5 = (s8<<30)|(s7>>2);
	temp6 = (s9<<17)| (s8>>15);

	t1= temp1^temp2;
	t2= temp3^temp4;
	t3= temp5^temp6;
	z1 = temp1^temp2^temp3^temp4^temp5^temp6;
	//z1=z;
	//����t1,t2,t3
	//t1 = t1 + s91s92 + s171
	temp1 = (s2<<5)|(s1>>27);
	temp2 = (s2<<4)|(s1>>28);
	temp3 = (s5<<18)|(s4>>14);

	t1 ^= (temp1&temp2)^temp3;

	//t2 = t2 + s175s176 + s264
	temp1 = (s5<<14)|(s4>>18);
	temp2 = (s5<<13)|(s4>>19);
	temp3 = (s8<<9)|(s7>>23);

	t2 ^= (temp1&temp2)^temp3;

	//t3 = t3 + s286s287 + s69
	temp1 = (s9<<19)|(s8>>13);
	temp2 = (s9<<18)|(s8>>14);
	temp3 = (s2<<27)|(s1>>5);

	t3 ^= (temp1&temp2)^temp3;

	// update register 1
	s2 = (s1)&(0x1FFFFFFF);
	s1 =s0;
	s0 = t3;

	//update register 2
	s5 =s4&(0x000FFFFF);
	s4 =s3;
	s3 = t1;

	//update register 3
	s9 =s8&(0x00007FFF);
	s8 =s7;
	s7 =s6;
	s6 = t2;
  }
	temp1 = (s2<<30)|(s1>>2);//���֮���൱��ֱ�Ӽ���
	temp2 = (s2<<3)|(s1>>29);
	temp3 = (s5<<27)|(s4>>5);
	temp4 = (s5<<12)|(s4>>20);
	temp5 = (s8<<30)|(s7>>2);
	temp6 = (s9<<17)| (s8>>15);
	z = temp1^temp2^temp3^temp4^temp5^temp6;
	if(roundnum_bit!=0)
		z=(z1<<roundnum_bit)|(z>>(32-roundnum_bit));
	//keystream[0]=reverse_word(z);
	temp1=0;
	temp1 = ((z&0x000000FF)<<24)^((z&0x0000FF00)<<8)^((z&0x00FF0000)>>8)^((z&0xFF000000)>>24);
	temp1 = ((temp1&0x01010101)<<7)^((temp1&0x02020202)<<5)^((temp1&0x04040404)<<3)^((temp1&0x08080808)<<1)^((temp1&0x10101010)>>1)^((temp1&0x20202020)>>3)^((temp1&0x40404040)>>5)^((temp1&0x80808080)>>7);
	keystream[0]=temp1;
}
 
 __global__ void genkeystream_thd(u32 cube[],u32 roundnum ,u32 *c,u32 loadkey[10],u32 offset,u32 dim,u32 k)
{
	//__shared__ u32 cache[1024];
	//__shared__ u32 cube_d[40],key_d[10];
	//__shared__ u32 iv[10];
	u32 i=0,j=0,iv[10];//loadkey[10]={0},cube[20]={1,2,3,4,5,6,7,8,9,10,11,12,13,14,15,16,17,18,19,20};
	u32 keystream[1];
	u32 tid;
	//i=k;
	for(j=0;j<10;j++)
		iv[j]=0;	
	j=k;
	for(i=0;i<4;i++)
	{
		tid=dim+i;
		iv[cube[tid]>>3]|=(j&0x01)<<(cube[tid]&0x07);
		j>>=1;
	}
	tid=threadIdx.x+blockIdx.x*blockDim.x+offset;//��ε���genkey,ÿ���߳�ֻ������512�������
	i=tid;
	for (j=0;j<dim;j++) 
		iv[cube[j]>>3] |= ( ( (i>>j) & 0x00000001 ) << (cube[j] & 0x07) );
	ECRYPT_keystream_wordsV3(keystream,iv,loadkey,roundnum);
	c[tid-offset]=keystream[0];//����32����
	__syncthreads();
}

  __global__ void genkeystream_thd_32(u32 cube[],u32 roundnum ,u32 *c,u32 loadkey[10],unsigned __int64 offset,u32 dim,u32 k)
{
	//__shared__ u32 cache[1024];
	//__shared__ u32 cube_d[40],key_d[10];
	//__shared__ u32 iv[10];
	unsigned __int64 i=0,j=0,l,m;
	u32 iv[10];//loadkey[10]={0},cube[20]={1,2,3,4,5,6,7,8,9,10,11,12,13,14,15,16,17,18,19,20};
	u32 keystream[1];
	unsigned __int64 tid;
	u32 streambit=0;
	//i=k;
	for(j=0;j<10;j++)
		iv[j]=0;	
	j=k;
	for(i=0;i<7;i++)
	{
		tid=dim+i;
		iv[cube[tid]>>3]|=(j&0x01)<<(cube[tid]&0x07);
		j>>=1;
	}
	tid=(threadIdx.x+blockIdx.x*blockDim.x+offset)*32;//��ε���genkey,ÿ���߳�ֻ������512�������
	i=tid;
	for(i=tid;i<tid+32;i++)//loopnumҪ����32���
	{
		for (j=0;j<dim;j++) 
			iv[cube[j]>>3] |= ( ( (i>>j) & 0x00000001 ) << (cube[j] & 0x07) );
		ECRYPT_keystream_wordsV3(keystream,iv,loadkey,roundnum);
		streambit|=(keystream[0]&0x01)<<(i-tid);//��������ڸ�λ
		for(j=0;j<10;j++)
			iv[j]=0;	
		j=k;
		for(l=0;l<7;l++)
		{
			m=dim+l;
			iv[cube[m]>>3]|=(j&0x01)<<(cube[m]&0x07);
			j>>=1;
		}
	}
	c[threadIdx.x+blockIdx.x*blockDim.x]=streambit;//����32����
	__syncthreads();
}

   __global__ void genkeystream_thd_128(u32 cube[],u32 roundnum ,u32 *c,u32 loadkey[10],u32 offset,u32 dim,u32 k)
{
	//__shared__ u32 cache[1024];
	//__shared__ u32 cube_d[40],key_d[10];
	//__shared__ u32 iv[10];
	unsigned __int64 i=0,j=0,l,m,n;
	u32 iv[10];//loadkey[10]={0},cube[20]={1,2,3,4,5,6,7,8,9,10,11,12,13,14,15,16,17,18,19,20};
	u32 keystream[1];
	unsigned __int64 tid;
	u32 streambit[4]={0};
	//i=k;
	for(j=0;j<10;j++)
		iv[j]=0;	
	j=k;
	for(i=0;i<6;i++)
	{
		tid=dim+i;
		iv[cube[tid]>>3]|=(j&0x01)<<(cube[tid]&0x07);
		j>>=1;
	}
	tid=(threadIdx.x+blockIdx.x*blockDim.x+offset)*128;//��ε���genkey,ÿ���߳�ֻ������512�������
	i=tid;
	for(n=0;n<4;n++)
	{
		tid=tid+n*32;
		streambit[n]=0;
		for(i=tid;i<tid+32;i++)//loopnumҪ����32���
		{
			for (j=0;j<dim;j++) 
				iv[cube[j]>>3] |= ( ( (i>>j) & 0x00000001 ) << (cube[j] & 0x07) );
			ECRYPT_keystream_wordsV3(keystream,iv,loadkey,roundnum);
			streambit[n]|=(keystream[0]&0x01)<<(i-tid);
			for(j=0;j<10;j++)
				iv[j]=0;	
			j=k;
			for(l=0;l<6;l++)
			{
				m=dim+l;
				iv[cube[m]>>3]|=(j&0x01)<<(cube[m]&0x07);
				j>>=1;
			}
		}
		//j=(threadIdx.x+blockIdx.x*blockDim.x)*4+m;
		//c[threadIdx.x+blockIdx.x*blockDim.x]=streambit;
		c[(threadIdx.x+blockIdx.x*blockDim.x)*4+n]=streambit[n];
	}
	/*c[(threadIdx.x+blockIdx.x*blockDim.x)*4+0]=streambit[0];
	c[(threadIdx.x+blockIdx.x*blockDim.x)*4+1]=streambit[1];
	c[(threadIdx.x+blockIdx.x*blockDim.x)*4+2]=streambit[2];
	c[(threadIdx.x+blockIdx.x*blockDim.x)*4+3]=streambit[3];*/
//	c[(threadIdx.x+blockIdx.x*blockDim.x)]=streambit;//����32����
	__syncthreads();
}
 //ֱ�ӽ�,����key������ȥ,Ȼ��ֱ�Ū�ͺ�
__global__  void sum_cube_word(u32  dim, u32 *cube,u32 roundnum ,u32 *c,u32 loadkey[10])
{
	__shared__ u32 cache[threadsPerBlock];
	unsigned __int64 i=0,j=0;
	u32 iv[10];
	u32 keystream[1];
	u32 keystream_sum=0;
//	loopnum= (unsigned __int64 ) pow((double)(2),(double)(dim));
	//loopnum= (unsigned __int64 ) pow(2.0,(double) (dim));
	i=threadIdx.x+blockIdx.x*blockDim.x;
	for(j=0;j<10;j++)
		iv[j]=0;	
	while(i<(U64C(0x01)<<dim))
	//while(i<loopnum)
	{
		for (j=0;j<dim;j++) 
			iv[cube[j]>>3] |= ( ( (i>>j) & 0x00000001 ) << (cube[j] & 0x07) );
		
		ECRYPT_keystream_wordsV3(keystream,iv,loadkey,roundnum);
		keystream_sum ^= keystream[0];
		//����iv
		for (j=0;j<10;j++) 
			iv[j]= 0;
		i+=blockDim.x*gridDim.x;
	}
	cache[threadIdx.x]=keystream_sum;
	__syncthreads();
	i=blockDim.x>>1;
	while(i!=0)
	{
		if(threadIdx.x<i)
			cache[threadIdx.x]^=cache[threadIdx.x+i];
		__syncthreads();
		i=i>>1;
	}
	if(threadIdx.x==0)
		c[blockIdx.x]=cache[0];
}


__global__  void sum_cube_word_32(u32  dim, u32 *cube,u32 roundnum ,u32 *c,u32 loadkey[10])
{
	__shared__ u32 cache[threadsPerBlock];
	unsigned __int64 i=0,j=0,k=0;
	u32 iv[10];
	u32 keystream[1];
	u32 keystream_sum=0;
//	loopnum= (unsigned __int64 ) pow((double)(2),(double)(dim));
	//loopnum= (unsigned __int64 ) pow(2.0,(double) (dim));
	i=(threadIdx.x+blockIdx.x*blockDim.x)*32;
	for(j=0;j<10;j++)
		iv[j]=0;	
	while(i<(U64C(0x01)<<dim))
	//while(i<loopnum)
	{

		///ÿ��һ�μ������32����
		for(k=i;k<i+32;k++)
		{
			
			for (j=0;j<dim;j++) 
				iv[cube[j]>>3] |= ( ( (k>>j) & 0x00000001 ) << (cube[j] & 0x07) );
		
			ECRYPT_keystream_wordsV3(keystream,iv,loadkey,roundnum);
			keystream_sum ^= keystream[0];
			//����iv
			for (j=0;j<10;j++) 
				iv[j]= 0;
		}

		i+=(blockDim.x*gridDim.x)*32;
	}
	cache[threadIdx.x]=keystream_sum;
	__syncthreads();
	i=blockDim.x>>1;
	while(i!=0)
	{
		if(threadIdx.x<i)
			cache[threadIdx.x]^=cache[threadIdx.x+i];
		__syncthreads();
		i=i>>1;
	}
	if(threadIdx.x==0)
		c[blockIdx.x]=cache[0];
}
//���β�ֵר��, kn��ʾ����һ����Կ,loc��ʾȡֵ
__global__  void sum_cube_word_sf(u32  dim, u32 *cube,u32 roundnum ,u32 *c,u32 loadkey[10], u32 kn, u32 loc)
{
	__shared__ u32 cache[threadsPerBlock];
	unsigned __int64 i=0,j=0;
	u32 iv[10],tempkey[10];
	u32 keystream[1];
	u32 keystream_sum=0;
//	loopnum= (unsigned __int64 ) pow((double)(2),(double)(dim));
	//loopnum= (unsigned __int64 ) pow(2.0,(double) (dim));

	//���� kn��loc����
	/*loadkey[(i)>>3]=loadkey[(i)>>3]&((0xFF)^(0x01)<<((i)&0x07));
	loadkey[(i+1)>>3]=loadkey[(i+1)>>3]&((0xFF)^(0x01)<<((i+1)&0x07));*/
	
	for(i=0;i<10;i++)
		tempkey[i]=loadkey[i];



	/*tempkey[kn>>3]=tempkey[kn>>3]&((0xFF)^((loc/2)<<(0x07&kn)));
	tempkey[(kn+1)>>3]=tempkey[(kn+1)>>3]&((0xFF)^((loc%2)<<(0x07&(kn+1))));*/
	//�Ȼָ���00��Ȼ���ٻ���

	tempkey[kn>>3]=tempkey[kn>>3]&((0xFF)^((0x01)<<(0x07&kn)));
	tempkey[(kn+1)>>3]=tempkey[(kn+1)>>3]&((0xFF)^((0x01)<<(0x07&(kn+1))));
	
	
	tempkey[kn>>3]=tempkey[kn>>3]|(((loc/2)<<(0x07&kn)));
	tempkey[(kn+1)>>3]=tempkey[(kn+1)>>3]|(((loc%2)<<(0x07&(kn+1))));


	i=threadIdx.x+blockIdx.x*blockDim.x;

	/*if(i==0)
	{
			for(j=0;j<10;j++)
					printf("%d,",tempkey[j]);
			printf("\n");
			printf("%d\n",loc);
	}*/
	for(j=0;j<10;j++)
		iv[j]=0;	
	while(i<(U64C(0x01)<<dim))
	//while(i<loopnum)
	{
		for (j=0;j<dim;j++) 
			iv[cube[j]>>3] |= ( ( (i>>j) & 0x00000001 ) << (cube[j] & 0x07) );
		
		ECRYPT_keystream_wordsV3(keystream,iv,tempkey,roundnum);
		keystream_sum ^= keystream[0];
		//����iv
		for (j=0;j<10;j++) 
			iv[j]= 0;
		i+=blockDim.x*gridDim.x;
	}

	cache[threadIdx.x]=keystream_sum;
	__syncthreads();
	i=blockDim.x>>1;
	while(i!=0)
	{
		if(threadIdx.x<i)
			cache[threadIdx.x]^=cache[threadIdx.x+i];
		__syncthreads();
		i=i>>1;
	}
	if(threadIdx.x==0)
		c[blockIdx.x]=cache[0];
}
__host__ u8 linearity_test_randomkey_word(u32 cube[], u8 dim, u32 roundnum,u8 linearity_test_res[])
{
	u32 chosenkey[48+1][10],i,keynum,partialsum[1024]={0},value[48]={0},j;
	//���鳤�Ȼ��д���ȶ,��Ҫ����ά������ȷ��
	//��Ҫ���ݵĲ���,�����Կ,cube,roundnum,����Ҫ������һ��ռ����ڴ�ŷ��صĲ������ֵ,�������Ͻ������,����һ������
	u32 **chosenkey_dev,*partialsum_dev;
	u32 blocksPerGrid= 64;//(0x01<<dim)/threadsPerBlock;
	u32 constantterm=0;
	u32 length,check=0,check_con=0;
	u32 *cube_dev,res,check_bit,flag;
	//size_t pitch;
	keynum=48;
	clock_t t1,t2;

	length=blocksPerGrid;//((0x01<<dim)>>4)>>8;
	///length=((0x01<<dim);
	/*printf("cube of dim %d is being tested \n",dim);
	for(i=0;i<dim;i++)
		printf("%d,",cube[i]);
	printf("\n");*/
	t1=clock_t();
	hipMalloc((void **)&cube_dev,dim*sizeof(u32));
	hipMalloc((void **)&partialsum_dev,length*sizeof(u32));
	chosenkey_dev=(u32 **)malloc(sizeof(u32 *)*(keynum+1));
	for(i=0;i<keynum+1;i++)
	{
		hipMalloc((void **) &chosenkey_dev[i],10*sizeof(u32));
		
	}

	for(i=1;i<keynum+1;i+=3)
	{
		choose_random_key(chosenkey[i]);
		choose_random_key(chosenkey[i+1]);
		for(j=0;j<10;j++)
			chosenkey[i+2][j]=chosenkey[i][j]^chosenkey[i+1][j];
	}
	for(i=0;i<10;i++)
		chosenkey[0][i]=0;
	//�������˵����ݿ������豸��
	hipMemcpy(cube_dev,cube,dim*sizeof(u32),hipMemcpyHostToDevice);
	for(i=0;i<keynum+1;i++)
	{
		hipMemcpy(chosenkey_dev[i],chosenkey[i],sizeof(u32)*10,hipMemcpyHostToDevice);
	}
	hipEvent_t start,stop;
	hipEventCreate(&start);  
    hipEventCreate(&stop);  
    hipEventRecord(start,0);  
//	sum_cube_word<<<blocksPerGrid,threadsPerBlock>>>(dim,cube_dev,roundnum,partialsum_dev,chosenkey_dev[0]);
	sum_cube_word_32<<<blocksPerGrid,threadsPerBlock>>>(dim,cube_dev,roundnum,partialsum_dev,chosenkey_dev[0]);
	//sum_cube_wordV2<<<blocksPerGrid,threadsPerBlock>>>(dim,cube_dev,roundnum,partialsum_dev,0);
	hipMemcpy(partialsum,partialsum_dev,sizeof(u32)*length,hipMemcpyDeviceToHost);
	hipEventRecord(stop,0);  
    hipEventSynchronize(stop);
	float tm;  
    hipEventElapsedTime(&tm,start,stop);  
    printf("GPU Elapsed time:%.6f ms.\n",tm);  
	for(i=0;i<length;i++)
		constantterm^=partialsum[i];


	//ʵ����Կ���á�
	for(j=0;j<keynum;j+=3)
	{
		//key1
		//sum_cube_word<<<blocksPerGrid,threadsPerBlock>>>(dim,cube_dev,roundnum,partialsum_dev,chosenkey_dev[j+1]);
		sum_cube_word_32<<<blocksPerGrid,threadsPerBlock>>>(dim,cube_dev,roundnum,partialsum_dev,chosenkey_dev[j+1]);
		hipMemcpy(partialsum,partialsum_dev,sizeof(u32)*length,hipMemcpyDeviceToHost);
		for(i=0;i<length;i++)
			value[j]^=partialsum[i];
		//key2
		//sum_cube_word<<<blocksPerGrid,threadsPerBlock>>>(dim,cube_dev,roundnum,partialsum_dev,chosenkey_dev[j+2]);
		sum_cube_word_32<<<blocksPerGrid,threadsPerBlock>>>(dim,cube_dev,roundnum,partialsum_dev,chosenkey_dev[j+2]);
		hipMemcpy(partialsum,partialsum_dev,sizeof(u32)*length,hipMemcpyDeviceToHost);
		for(i=0;i<length;i++)
			value[j+1]^=partialsum[i];
		//key1+key2
		//sum_cube_word<<<blocksPerGrid,threadsPerBlock>>>(dim,cube_dev,roundnum,partialsum_dev,chosenkey_dev[j+3]);
		sum_cube_word_32<<<blocksPerGrid,threadsPerBlock>>>(dim,cube_dev,roundnum,partialsum_dev,chosenkey_dev[j+3]);
		hipMemcpy(partialsum,partialsum_dev,sizeof(u32)*length,hipMemcpyDeviceToHost);
		for(i=0;i<length;i++)
			value[j+2]^=partialsum[i];
		//
		check_con = check_con| (constantterm^value[j]);
		check = check | (constantterm^value[j]^value[j+1]^value[j+2]);
		if(check == 0xFFFFFFFF)
			break;
	}
	if(check==0xFFFFFFFF)//Ҫ�ĳ�FFFFFFFF����������
	{
		for(i=0;i<32;i++)
			linearity_test_res[i] = 0;
		/*for(i=0;i<32;i++)
			printf("%d,",linearity_test_res[i]);
		printf("\n");*/
		return 0;
	}
	else{
		res = 2;
		for(i=0;i<32;i++)
		{
			check_bit = (check>>i) & 0x01;
			if(check_bit==0x01)
				linearity_test_res[i]=0;
			else
			{
				flag = 0;
				//����Ƿ�ֵ
				if((check_con>>i) & 0x01)
					linearity_test_res[i] = 1;
				else
					linearity_test_res[i] = 2;
			}	
		}
	}
	/*for(i=0;i<32;i++)
			printf("%d,",linearity_test_res[i]);
	printf("\n");*/
	//u32 **chosenkey_dev,*partialsum_dev;
	hipFree(cube_dev);
	hipFree(partialsum_dev);
	for(i=0;i<keynum+1;i++)
		hipFree(chosenkey_dev[i]);
	//cudaEvent
	hipEventDestroy(start);
	hipEventDestroy(stop);
	return res;
}

__host__ void RetrieveSum(u32 cube[], u8 dim, u32 roundnum,u32 chosenkey[][10], u32 sum_res[])
{
	u32 i,keynum,partialsum[1024]={0},value[100]={0},j;
	//���鳤�Ȼ��д���ȶ,��Ҫ����ά������ȷ��
	//��Ҫ���ݵĲ���,�����Կ,cube,roundnum,����Ҫ������һ��ռ����ڴ�ŷ��صĲ������ֵ,�������Ͻ������,����һ������
	u32 **chosenkey_dev,*partialsum_dev;
	u32 blocksPerGrid= 64;//(0x01<<dim)/threadsPerBlock;
	u32 constantterm=0;
	u32 length,check=0,check_con=0;
	u32 *cube_dev,res,check_bit,flag;
	//size_t pitch;
	keynum=20;
	clock_t t1,t2;

	length=blocksPerGrid;//((0x01<<dim)>>4)>>8;

	t1=clock_t();
	hipMalloc((void **)&cube_dev,dim*sizeof(u32));
	hipMalloc((void **)&partialsum_dev,length*sizeof(u32));
	chosenkey_dev=(u32 **)malloc(sizeof(u32 *)*(keynum));
	for(i=0;i<keynum;i++)
	{
		hipMalloc((void **) &chosenkey_dev[i],10*sizeof(u32));
		
	}
	//�������˵����ݿ������豸��
	hipMemcpy(cube_dev,cube,dim*sizeof(u32),hipMemcpyHostToDevice);
	for(i=0;i<keynum;i++)
	{
		hipMemcpy(chosenkey_dev[i],chosenkey[i],sizeof(u32)*10,hipMemcpyHostToDevice);
	}

	//ʵ����Կ���á�
	for(j=0;j<keynum;j++)
	{
		//key j
		sum_cube_word_32<<<blocksPerGrid,threadsPerBlock>>>(dim,cube_dev,roundnum,partialsum_dev,chosenkey_dev[j]);
		hipMemcpy(partialsum,partialsum_dev,sizeof(u32)*length,hipMemcpyDeviceToHost);
		for(i=0;i<length;i++)
			value[j]^=partialsum[i];

		sum_res[j]=value[j]&0x01;
		
	}
	
	hipFree(cube_dev);
	hipFree(partialsum_dev);
	for(i=0;i<keynum+1;i++)
		hipFree(chosenkey_dev[i]);
}

__host__ void GetSum()
{
	//cubeҲ��������
	//�������
	u32 roundnum=709, randomkey[1000][10];
	u32 keynum=20;
	u8 lin_res[32],flag;
	u32 cube[100][80]={{73,71,70,66,57,55,51,49,47,43,40,39,36,34,32,31,19,16,12,11,6,3,0},{71,32,9,66,2,70,11,59,72,20,56,41,58,43,77,5,74,25,6,47,33,68,17,36},{33,71,32,63,40,37,4,30,5,76,7,50,73,3,15,12,41,59,47,2,51,35,56,65,22},
{12,43,59,70,61,42,66,64,48,40,35,55,19,67,77,23,54,17,72,63,7,79,4,57},
{33,71,32,63,40,37,4,30,5,76,7,50,9,73,3,15,12,59,47,2,35,56,65,22,27,31},
{75,71,67,35,47,55,4,5,62,32,17,65,50,37,58,48,26,0,23,33,77,20,38,60,7,79,14,64},
{75,74,71,67,35,47,55,5,62,32,17,50,37,58,48,0,23,77,20,60,7,79,14,41,64}};
	u32 i,j,k,recsum[1000];
	u32 cubenum=5;
	u8 dimlist[100]={23,24,25,25,26,28,25};
	FILE *RecKey, *RecSum,*fp;
	RecKey=fopen("RecKey","a+");
	RecSum=fopen("RecSum","a+");
	// record the chosen key
	for(i=0;i<keynum;i++)
	{
		choose_random_key(randomkey[i]);
		for(j=0;j<10;j++)
			fprintf(RecKey,"%x,", randomkey[i][j]);
		fprintf(RecKey,"\n");
	}
	for(i=0;i<1000;i++)
	{
		for(j=0;j<10;j++)
			randomkey[i][j]=0x00;
		randomkey[i][2]=0x02;
	}
	// get sum for each cube

	for(i=0;i<cubenum;i++)
	{
		// compute the sum of the chosen 100 random keys
		printf("%d,",i);
		//
		flag=linearity_test_randomkey_word(cube[i],dimlist[i],roundnum,lin_res);
		printf("*******%d*******",flag);
		RetrieveSum(cube[i],dimlist[i],roundnum,randomkey,recsum);

		for(j=0;j<dimlist[i];j++)
		{
			fprintf(RecSum,"%u,",cube[i][j]);
		}
		fprintf(RecSum,"   ");
		for(j=0;j<keynum;j++)
			fprintf(RecSum,"%d,",recsum[j]);
		fprintf(RecSum,"\n");
	}
	fclose(RecSum);
	fclose(RecKey);
}

__host__ void verifygivencube(u32 cube[], u8 dim, u32 roundnum, u8 outindex)
{
	u8 i,result=0x00;
	u8 linearity_test_res[32];
	
	clock_t t1,t2;
	for(i=0;i<dim;i++)
		printf("%d,",cube[i]);
	printf("\n");
	t1=clock_t();
	
	result=linearity_test_randomkey_word(cube, dim, roundnum,linearity_test_res);

	t2=clock_t();
	printf("%d ms\n",t2-t1);
	for(i=0;i<32;i++)
		printf("%d,",linearity_test_res[i]);
	printf("\n");
	if(linearity_test_res[outindex]==1){
		printf("superpoly is linear! \n");
			//printfsuperpoly(cube, dim,roundnum, outindex);
		//printfsuperpoly(cube,dim,roundnum,outindex);
	}
	else if(linearity_test_res[outindex] ==2)
		printf("superpoly is a constant! \n");
	else
		printf("superpoly is nonlinear!\n");
}
__host__ void write_superpoly_subcube(FILE * fp,u32 cube[], u8 dim,u32 roundnum, u8 outindex)
{
	u32 chosenkey[80+1][10],i,keynum,partialsum[1024]={0},value[81]={0},j;
	//���鳤�Ȼ��д���ȶ,��Ҫ����ά������ȷ��
	//��Ҫ���ݵĲ���,�����Կ,cube,roundnum,����Ҫ������һ��ռ����ڴ�ŷ��صĲ������ֵ,�������Ͻ������,����һ������
	u32 **randkey_dev,*psum_dev;
	u32 blocksPerGrid= 64;//(0x01<<dim)/threadsPerBlock;
	u32 constantterm=0;
	u32 length;
	u32 *cube_d;
	u8 coeff[81]={0};
	//size_t pitch;
	keynum=80;
	length=blocksPerGrid;//((0x01<<dim)>>4)>>8;
	///length=((0x01<<dim);
	hipMalloc((void **)&cube_d,dim*sizeof(u32));
	hipMalloc((void **)&psum_dev,length*sizeof(u32));
	randkey_dev=(u32 **)malloc(sizeof(u32 *)*(keynum+1));
	for(i=0;i<keynum+1;i++)
	{
		hipMalloc((void **) &randkey_dev[i],10*sizeof(u32));
		
	}

	for(i=1;i<keynum+1;i++)
	{
		for(j=0;j<10;j++)
			chosenkey[i][j]=0;
		chosenkey[i][(i-1)>>3]=(0x01)<<((i-1)&0x07);
	}
	for(i=0;i<10;i++)
		chosenkey[0][i]=0;
	//�������˵����ݿ������豸��
	hipMemcpy(cube_d,cube,dim*sizeof(u32),hipMemcpyHostToDevice);
	for(i=0;i<keynum+1;i++)
	{
		hipMemcpy(randkey_dev[i],chosenkey[i],sizeof(u32)*10,hipMemcpyHostToDevice);
	}
	hipEvent_t start,stop;
	hipEventCreate(&start);  
    hipEventCreate(&stop);  
    hipEventRecord(start,0);  
	sum_cube_word<<<blocksPerGrid,threadsPerBlock>>>(dim,cube_d,roundnum,psum_dev,randkey_dev[0]);
	//sum_cube_word_32<<<blocksPerGrid,threadsPerBlock>>>(dim,cube_d,roundnum,psum_dev,randkey_dev[0]);
	//sum_cube_wordV2<<<blocksPerGrid,threadsPerBlock>>>(dim,cube_d,roundnum,psum_dev,0);
	hipEventRecord(stop,0);  
    hipEventSynchronize(stop);  
    float tm;  
    hipEventElapsedTime(&tm,start,stop);  
    printf("GPU Elapsed time:%.6f ms.\n",tm);  

	hipMemcpy(partialsum,psum_dev,sizeof(u32)*length,hipMemcpyDeviceToHost);
	for(i=0;i<length;i++)
		constantterm^=partialsum[i];
	coeff[0]=(constantterm>>outindex)&(0x01);
	fprintf(fp,"%d",coeff[0]);
	for(j=1;j<keynum+1;j++)
	{
		sum_cube_word<<<blocksPerGrid,threadsPerBlock>>>(dim,cube_d,roundnum,psum_dev,randkey_dev[j]);
		hipMemcpy(partialsum,psum_dev,sizeof(u32)*length,hipMemcpyDeviceToHost);
		for(i=0;i<length;i++)
			value[j]^=partialsum[i];
		coeff[j]=((value[j]>>outindex)&0x01)^coeff[0];
		if(coeff[j]==1)
			fprintf(fp,"+x%d",j-1);
		
	}
	 fprintf(fp, "\n");
	//u32 **randkey_dev,*psum_dev;
	 hipFree(cube_d);
	hipFree(psum_dev);
	for(i=0;i<keynum+1;i++)
		hipFree(randkey_dev[i]);
	free(randkey_dev);
	hipEventDestroy(start);
	hipEventDestroy(stop);
}

//u8 outputsubcubes(u32 cube[], u8 dim, u32 roundnum, u8 outindex, u32 pos, char filename[])
u8 outputsubcubes(u32 cube[], u8 dim, u32 roundnum, u8 outindex, unsigned __int64 pos, char filename[])
{
	u8 i,j;
	u32 *subcube;
	u8 subdim;
	u8 result=0x00;
	u8 linearity_test_res[32];
	FILE * fp;
	subcube = (u32 *)malloc(dim*sizeof(u32));

	fp = fopen(filename,"a+");
	subdim = 0;
	for (j=0;j<dim;j++) {
		if((pos>>j) & 0x00000001)
		{
			subcube[subdim] = cube[j];
			subdim++;
		}
	}	

	//result=linearity_test_randomkey_word(subcube, subdim, roundnum,linearity_test_res);
	//result=1;
	//linearity_test_res[outindex]=1;
	//if(linearity_test_res[outindex]==1)
	if(1)
	{
		if(firstoutput)
		{
			fprintf(fp, "\n\nround = %d*32, dim = %d,",roundnum,dim);
			fprintf(fp, "\nmother cube: ");
			for(i=0;i<dim;i++)
				fprintf(fp, "%d,",cube[i]);
			firstoutput = 0;
		}
		fprintf(fp, "\noutindex = %d, subdim = %d", outindex,subdim);
		fprintf(fp,"\nTYPE1: subcube: ");
		for(j=0;j<subdim;j++)
			fprintf(fp,"%d,",subcube[j]);
		fprintf(fp,"	");
		write_superpoly_subcube(fp,subcube, subdim,roundnum, outindex);
		fclose(fp);
		free(subcube);
		return 1;
	}
	else
	{
		fclose(fp);
		free(subcube);
		return 0;
	}
/*	else if(linearity_test_res[outindex] ==2)
	{
		if(firstoutput)
		{
			fprintf(fp, "\n\nround = %d*32, dim = %d,",roundnum,dim);
			fprintf(fp, "\nmother cube: ");
			for(i=0;i<dim;i++)
				fprintf(fp, "%d,",cube[i]);
			firstoutput = 0;
		}
		fprintf(fp, "\noutindex = %d, subdim = %d", outindex,subdim);
		fprintf(fp,"\nsubcube: ");
		for(j=0;j<subdim;j++)
			fprintf(fp,"%d,",subcube[j]);
		fprintf(fp,"	");
		fprintf(fp,"constant");
	}*/

}



 __host__ void construc_truth_table_dynamicV2(u32 cube[], u32 dim, u32 roundnum,u32 **table,u32 rownum, u32 columnnum,u32 loadkey[10])
{
	////ÿһ���߳̿���ֻ��һ���߳�,Ȼ��ÿһ���̲߳���512�������Կ��,��ѭ���Ĵ�������2^dim/512/�߳̿�ĸ���

	u32 i=0,j=0,k=0,iv[10]={0};
	//u32 keystream[1];
	u32 temp=0,*partialstream_dev,*partialstream,*loadkey_dev,*iv_dev;//partialstream���ڴ洢���豸�˲�������Կ������
	u32 bolcknum=64,threadnum=512,loopnum;
	//dim3 bolckdim;
	u32 *cube_dev;
	for(i=0;i<rownum;i++)
		for(j=0;j<columnnum;j++)
			table[i][j] = 0;

	//ȷ����ѭ������
	loopnum=(U64C(0x01)<<dim)/bolcknum/threadnum;
	//����ռ�
	partialstream=(u32 *)malloc(sizeof(u32 )*bolcknum*threadnum);
	hipMalloc((void **)&partialstream_dev,sizeof(u32 )*bolcknum*threadnum);
	hipMalloc((void **)&cube_dev,sizeof(u32)*dim);
	hipMalloc((void **)&loadkey_dev,sizeof(u32 )*10);
	hipMalloc((void **)&iv_dev,sizeof(u32 )*10);
	//hipMallocPitch((void**)&cube_dev,&pitch_cube,sizeof(u8)*dim,1);
	//hipMallocPitch((void**)&loadkey_dev,&pitch_key,sizeof(u32)*10,1);
	//��������
	
	hipMemcpy(cube_dev,cube,sizeof(u32)*dim,hipMemcpyHostToDevice);
	hipMemcpy(loadkey_dev,loadkey,sizeof(u32)*10,hipMemcpyHostToDevice);
	hipMemcpy(iv_dev,iv,sizeof(u32)*10,hipMemcpyHostToDevice);
	//hipMemcpy2D(cube_dev,pitch_cube,cube,sizeof(u8)*dim,sizeof(u8)*dim,1,hipMemcpyHostToDevice);
	//hipMemcpy2D(loadkey_dev,pitch_key,loadkey,sizeof(u32)*10,sizeof(u32)*10,1,hipMemcpyHostToDevice);
	hipEvent_t start,stop;
	hipEventCreate(&start);  
    hipEventCreate(&stop);  
    float tm;  
  
	/*�ø�����cube������Կ��*/
	//���õĿ鲢��,�����̲߳��еĻ�
	for (i=0;i<loopnum;i++)
	{
		//sum_cube_word<<<blocksPerGrid,threadsPerBlock>>>(dim,cube_dev,roundnum,partialsum_dev,chosenkey_dev[j+3]);
		temp=i*bolcknum*threadnum;//ƫ����
		hipEventRecord(start,0); 
		//genkeystream_thd<<<bolcknum,threadnum>>>(cube_dev,roundnum,partialstream_dev,loadkey_dev,temp,dim);
		hipEventRecord(stop,0);  
		hipEventSynchronize(stop); 
		hipEventElapsedTime(&tm,start,stop);  
	//	printf("GPU Elapsed time:%.6f ms.\n",tm);
		hipMemcpy(partialstream,partialstream_dev,sizeof(u32)*bolcknum*threadnum,hipMemcpyDeviceToHost);//�������һ������32�ֵĻ�Ӧ�û�����
		//
		//
		//for(k=0;k<bolcknum*threadnum;k++)
		for(k=0;k<bolcknum*threadnum;k++)
		{
			temp=i*bolcknum*threadnum+k;//0-256,ÿһ������1024�������,�����Ż�Ч������
		for(j=0;j<1;j++)
			table[j][temp>>5] |= ((partialstream[k]>>j)&0x00000001)<<(temp&0x0000001F);
		}
	}
}

u32 linearity_test_dynamicV2(u32 cube[], u32 dim, u32 roundnum,char filename[])
{
	u32 i,j,a,b,key[10];
	u32 **constantterm;//������
	u32 numrandomkey = 16*2;  // ����һ��
	u32 randomkey[16*2][10];
	u32 **value0;//ÿ�������Կ�ĺ���ֵ
	u32 **value1;//ÿ�������Կ�ĺ���ֵ
	u32 **twokeysum; //���������Կ���ĺ���ֵ
	u32 **ANF,**ANF2;
	u32 **check,**check1;
	FILE *fp2,*Sdim;
	u32 flag,flag1,subdim;
	u32 rownum = 32,cubenum=0;
	u32 columnum = U64C(0x01)<<(dim-5);
	clock_t t1,t2;
	fp2=fopen("CRec.txt","a+");
	Sdim=fopen("subdim.txt","a+");
	constantterm = (u32 **)malloc(rownum*sizeof(u32*));
	for(i=0;i<rownum;i++)
		constantterm[i]=(u32*)malloc(columnum*sizeof(u32));

	value0 = (u32 **)malloc(rownum*sizeof(u32*));
	for(i=0;i<rownum;i++)
		value0[i]=(u32*)malloc(columnum*sizeof(u32));

	value1 = (u32 **)malloc(rownum*sizeof(u32*));
	for(i=0;i<rownum;i++)
		value1[i]=(u32*)malloc(columnum*sizeof(u32));

	twokeysum = (u32 **)malloc(rownum*sizeof(u32*));
	for(i=0;i<rownum;i++)
		twokeysum[i]=(u32*)malloc(columnum*sizeof(u32));

	ANF = (u32 **)malloc(rownum*sizeof(u32*));
	ANF2 = (u32 **)malloc(rownum*sizeof(u32*));
	for(i=0;i<rownum;i++)
	{
		ANF[i]=(u32*)malloc(columnum*sizeof(u32));
		ANF2[i]=(u32*)malloc(columnum*sizeof(u32));
	}

	check = (u32 **)malloc(rownum*sizeof(u32*));
	check1 = (u32 **)malloc(rownum*sizeof(u32*));
	for(i=0;i<rownum;i++)
	{
		check[i]=(u32*)malloc(columnum*sizeof(u32));
		check1[i]=(u32*)malloc(columnum*sizeof(u32));
	}

	for(a=0;a<rownum;a++)
		for(b=0;b<columnum;b++)
		{
				check[a][b] = 0;
				check1[a][b] = 0;
		}

	for(i=0;i<10;i++)
		key[i]= 0;
	//ECRYPT_keysetup(&ctx, key, 80,80);
	construc_truth_table_dynamicV2(cube, dim, roundnum,constantterm,rownum,columnum,key);

	for(i=0;i<numrandomkey;i++)
		choose_random_key(randomkey[i]);

	for(j=0;j<numrandomkey;j=j+2)
	{
		for(i=0;i<10;i++)
			key[i] = randomkey[j][i];
		//ECRYPT_keysetup(&ctx, key, 80,80);
		construc_truth_table_dynamicV2(cube, dim, roundnum,value0,rownum,columnum,key);

		for(i=0;i<10;i++)
			key[i]=randomkey[j+1][i];
		//ECRYPT_keysetup(&ctx, key, 80,80);
		construc_truth_table_dynamicV2(cube, dim, roundnum,value1,rownum,columnum,key);

		for(i=0;i<10;i++)
			key[i]=randomkey[j][i]^randomkey[j+1][i];
		//ECRYPT_keysetup(&ctx, key, 80,80);
		t1=clock();
		construc_truth_table_dynamicV2(cube, dim, roundnum,twokeysum,rownum,columnum,key);
		t2=clock();
		printf("%dms\n",t2-t1);
		for(a=0;a<rownum;a++){
			for(b=0;b<columnum;b++)
			{
				ANF[a][b] = constantterm[a][b]^value0[a][b]^value1[a][b]^twokeysum[a][b];
				ANF2[a][b] = constantterm[a][b]^value0[a][b];
			}
			Moebius(ANF[a], U64C(0x01)<<dim);
			Moebius(ANF2[a], U64C(0x01)<<dim);
			for(b=0;b<columnum;b++)
			{
				check[a][b] |= ANF[a][b];
				check1[a][b] |= ANF2[a][b];
			}
		}
	}
	fprintf(fp2,"*************************\n");
	for(a=0;a<rownum;a++)
	{
		fprintf(fp2,"\nOutputBit %d is being tested.....\n",a);
		for(b=0;b<(U64C(0x01)<<dim);b++)
		{
			flag = check[a][b>>5] & ((U64C(0x01))<<(b&0x0000001F));
			flag1 = check1[a][b>>5] & ((U64C(0x01))<<(b&0x0000001F));
			//flag1=1;
			subdim=0;
			if((flag==0)&&(flag1!=0))//ͨ�����Լ��,���ǲ�ͨ����ֵ����
			{
				for(i=0;i<dim;i++)
				{
					if(((b>>i)&0x01) ==1)
					{
						fprintf(fp2,"%d,",cube[i]);
						subdim++;
					}
				}
				fprintf(fp2,"\n");
				fprintf(Sdim,"%d,",subdim);
				outputsubcubes(cube,dim,roundnum,a,b,filename);
			}
		}

	//printf("OutputBit %d is being tested.....",a);
	}
	
	for(i=0;i<rownum;i++)
	{
		free(constantterm[i]);
		free(value0[i]);
		free(value1[i]);
		free(twokeysum[i]);
		free(ANF[i]);
		free(check[i]);
	}
	free(constantterm);
	free(value0);
	free(value1);
	free(twokeysum);
	free(ANF);
	free(check);
	fclose(fp2);
	fclose(Sdim);
	return cubenum;
}

//cut memory
 __host__ void construc_truth_table_dynamic_CutM(u32 cube[], u32 dim, u32 roundnum,u32 **table,u32 rownum, u32 columnnum,u32 loadkey[10],u32 lk)
{
	////ÿһ���߳̿���ֻ��һ���߳�,Ȼ��ÿһ���̲߳���512�������Կ��,��ѭ���Ĵ�������2^dim/512/�߳̿�ĸ���

	u32 i=0,j=0,k=0,iv[10]={0};
	//u32 keystream[1];
	u32 temp=0,*partialstream_dev,*partialstream,*loadkey_dev,*iv_dev;//partialstream���ڴ洢���豸�˲�������Կ������
	u32 bolcknum=64,threadnum=512,loopnum;
	//dim3 bolckdim;
	u32 *cube_dev;
	//for()

	//ȷ����ѭ������
	loopnum=(U64C(0x01)<<(dim-6))/bolcknum/threadnum;
	//����ռ�
	partialstream=(u32 *)malloc(sizeof(u32 )*bolcknum*threadnum);
	hipMalloc((void **)&partialstream_dev,sizeof(u32 )*bolcknum*threadnum);
	hipMalloc((void **)&cube_dev,sizeof(u32)*dim);
	hipMalloc((void **)&loadkey_dev,sizeof(u32 )*10);
	hipMalloc((void **)&iv_dev,sizeof(u32 )*10);
	//hipMallocPitch((void**)&cube_dev,&pitch_cube,sizeof(u8)*dim,1);
	//hipMallocPitch((void**)&loadkey_dev,&pitch_key,sizeof(u32)*10,1);
	//��������
	
	hipMemcpy(cube_dev,cube,sizeof(u32)*dim,hipMemcpyHostToDevice);
	hipMemcpy(loadkey_dev,loadkey,sizeof(u32)*10,hipMemcpyHostToDevice);
	hipMemcpy(iv_dev,iv,sizeof(u32)*10,hipMemcpyHostToDevice);
	//hipMemcpy2D(cube_dev,pitch_cube,cube,sizeof(u8)*dim,sizeof(u8)*dim,1,hipMemcpyHostToDevice);
	//hipMemcpy2D(loadkey_dev,pitch_key,loadkey,sizeof(u32)*10,sizeof(u32)*10,1,hipMemcpyHostToDevice);
	hipEvent_t start,stop;
	hipEventCreate(&start);  
    hipEventCreate(&stop);  
    float tm;  
  
	/*�ø�����cube������Կ��*/
	//���õĿ鲢��,�����̲߳��еĻ�
	dim=dim-4;
	for (i=0;i<loopnum;i++)
	{
		//sum_cube_word<<<blocksPerGrid,threadsPerBlock>>>(dim,cube_dev,roundnum,partialsum_dev,chosenkey_dev[j+3]);
		temp=i*bolcknum*threadnum;//ƫ����
		hipEventRecord(start,0); 
		genkeystream_thd<<<bolcknum,threadnum>>>(cube_dev,roundnum,partialstream_dev,loadkey_dev,temp,dim,lk);
		hipEventRecord(stop,0);  
		hipEventSynchronize(stop); 
		hipEventElapsedTime(&tm,start,stop);  
		//
		hipMemcpy(partialstream,partialstream_dev,sizeof(u32)*bolcknum*threadnum,hipMemcpyDeviceToHost);
	//	printf("GPU Elapsed time:%.6f ms.\n",tm);
		//
		for(k=0;k<bolcknum*threadnum;k++)
		{
			temp=i*bolcknum*threadnum+k;//0-256,ÿһ������1024�������,�����Ż�Ч������
		for(j=0;j<1;j++)
			table[j][temp>>5] ^= ((partialstream[k]>>j)&0x00000001)<<(temp&0x0000001F);
		}
	}
}


  __host__ void construc_truth_table_dynamic_CutM_32(u32 cube[], u32 dim, u32 roundnum,u32 **table,u32 rownum, u32 columnnum,u32 loadkey[10],u32 lk)
{
	////ÿһ���߳̿���ֻ��һ���߳�,Ȼ��ÿһ���̲߳���512�������Կ��,��ѭ���Ĵ�������2^dim/512/�߳̿�ĸ���

	unsigned __int64 i=0,j=0,k=0, temp=0;
	u32 iv[10]={0};
	//u32 keystream[1];
	u32 *partialstream_dev,*partialstream,*loadkey_dev,*iv_dev;//partialstream���ڴ洢���豸�˲�������Կ������
	u32 bolcknum=64,threadnum=512;//00.06
	//u32 bolcknum=8,threadnum=32;
	unsigned __int64 loopnum;
	//dim3 bolckdim;
	u32 *cube_dev;
	//for()

	//ȷ����ѭ������
	loopnum=(U64C(0x01)<<(dim-7))/bolcknum/threadnum/32;
	//����ռ�
	partialstream=(u32 *)malloc(sizeof(u32 )*bolcknum*threadnum);
	hipMalloc((void **)&partialstream_dev,sizeof(u32 )*bolcknum*threadnum);
	hipMalloc((void **)&cube_dev,sizeof(u32)*dim);
	hipMalloc((void **)&loadkey_dev,sizeof(u32 )*10);
	hipMalloc((void **)&iv_dev,sizeof(u32 )*10);
	//hipMallocPitch((void**)&cube_dev,&pitch_cube,sizeof(u8)*dim,1);
	//hipMallocPitch((void**)&loadkey_dev,&pitch_key,sizeof(u32)*10,1);
	//��������
	
	hipMemcpy(cube_dev,cube,sizeof(u32)*dim,hipMemcpyHostToDevice);
	hipMemcpy(loadkey_dev,loadkey,sizeof(u32)*10,hipMemcpyHostToDevice);
	hipMemcpy(iv_dev,iv,sizeof(u32)*10,hipMemcpyHostToDevice);
	//hipMemcpy2D(cube_dev,pitch_cube,cube,sizeof(u8)*dim,sizeof(u8)*dim,1,hipMemcpyHostToDevice);
	//hipMemcpy2D(loadkey_dev,pitch_key,loadkey,sizeof(u32)*10,sizeof(u32)*10,1,hipMemcpyHostToDevice);
	//hipEvent_t start,stop;
//	hipEventCreate(&start);  
  //  hipEventCreate(&stop);  
    float tm;  
  
	/*�ø�����cube������Կ��*/
	//���õĿ鲢��,�����̲߳��еĻ�
	dim=dim-7;
	for (i=0;i<loopnum;i++)
	{
		//sum_cube_word<<<blocksPerGrid,threadsPerBlock>>>(dim,cube_dev,roundnum,partialsum_dev,chosenkey_dev[j+3]);
		temp=i*bolcknum*threadnum;//ƫ����
	//	hipEventRecord(start,0); 
		genkeystream_thd_32<<<bolcknum,threadnum>>>(cube_dev,roundnum,partialstream_dev,loadkey_dev,temp,dim,lk);
	//	genkeystream_thd_128<<<bolcknum,threadnum>>>(cube_dev,roundnum,partialstream_dev,loadkey_dev,temp,dim,lk);//
	//	hipEventRecord(stop,0);  
	//	hipEventSynchronize(stop); 
//		hipEventElapsedTime(&tm,start,stop);  
		//
		hipMemcpy(partialstream,partialstream_dev,sizeof(u32)*bolcknum*threadnum,hipMemcpyDeviceToHost);
	//	printf("GPU Elapsed time:%.6f ms.\n",tm);
		//
		for(k=0;k<bolcknum*threadnum;k++)
		{
			temp=i*bolcknum*threadnum+k;//0-256,ÿһ������1024�������,�����Ż�Ч������
		for(j=0;j<1;j++)
			//table[j][temp>>5] ^= ((partialstream[k]>>j)&0x00000001)<<(temp&0x0000001F);
			table[j][temp]^=partialstream[k];
		}
	}
	hipFree(cube_dev);
	hipFree(partialstream_dev);
	//hipFree(partialstream);
	hipFree(iv_dev);
	hipFree(loadkey);
	free(partialstream);
}
 ///singleRound
u32 linearity_test_dynamicV3(u32 cube[], u32 dim, u32 roundnum,char filename[])
{
	u32 i,j,a,b,key[10],k,partdim,l,n;
	u32 **constantterm,**constantterm1,**check1,**value0;//������
	u32 *Filter1,*Filter, Flength=32768, *Fcheck,*Fcheck1,*Fpos;
	u32 numrandomkey = 16*2;  // ����һ��
	u32 randomkey[16*2][10];
	u32 **check,**ANF;//������Ҫ�ų�������
	FILE *fp2,*Sdim;
	u32 flag,flag1,subdim;
	u32 rownum = 1,cubenum=0;
	u32 columnum = U64C(0x01)<<(dim-5);
	fp2=fopen("CRec.txt","a+");
	Sdim=fopen("subdim.txt","a+");
	n=Flength;
	constantterm = (u32 **)malloc(rownum*sizeof(u32*));
	Filter1=(u32 *) malloc(sizeof(u32)*Flength);
	Filter=(u32 *) malloc(sizeof(u32)*Flength);
	Fcheck=(u32 *) malloc(sizeof(u32)*Flength);
	Fcheck1=(u32 *) malloc(sizeof(u32)*Flength);
	Fpos=(u32 *) malloc(sizeof(u32)*Flength);
	for(i=0;i<rownum;i++)
		constantterm[i]=(u32*)malloc(columnum*sizeof(u32));

	constantterm1 = (u32 **)malloc(rownum*sizeof(u32*));
	for(i=0;i<rownum;i++)
		constantterm1[i]=(u32*)malloc(columnum*sizeof(u32));
	
	check1 = (u32 **)malloc(rownum*sizeof(u32*));
	value0 = (u32 **)malloc(rownum*sizeof(u32*));
	ANF = (u32 **)malloc(rownum*sizeof(u32*));
	for(i=0;i<rownum;i++)
	{
		ANF[i]=(u32*)malloc(columnum*sizeof(u32));
		check1[i]=(u32*)malloc(columnum*sizeof(u32));
		value0[i]=(u32*)malloc(columnum*sizeof(u32));
	}

	check = (u32 **)malloc(rownum*sizeof(u32*));
	for(i=0;i<rownum;i++)
	{
		check[i]=(u32*)malloc(columnum*sizeof(u32));
	}

	for(a=0;a<rownum;a++)
		for(b=0;b<columnum;b++)
		{
				check[a][b] = 0;
				check1[a][b] = 0;
		}

	//�������һ��ѭ��
	//partdim=4;
	partdim=4;
	for(i=0;i<numrandomkey;i++)
		choose_random_key(randomkey[i]);
	//dim-=partdim;
	for(k=0;k<Flength;k++)
	{
		Fcheck[k]=0;
		Fcheck1[k]=0;
		Filter[k]=0;
		Filter1[k]=0;
	}
	for(j=0;j<numrandomkey;j=j+2)
	{
		Flength=0;
		for(k=0;k<16;k++)
		{
			//ȡcube,Ȼ����㳣ֵ
			if((k&0x01)==0)
			{
				for(a=0;a<rownum;a++)
					for(b=0;b<columnum;b++)
					{
							constantterm[a][b] = 0;
							value0[a][b] = 0;
							constantterm1[a][b] = 0;
					}
			}
			for(i=0;i<10;i++)
				key[i]= 0;
			construc_truth_table_dynamic_CutM(cube, dim, roundnum,constantterm,rownum,columnum,key,k);
			for(i=0;i<10;i++)
				key[i] = randomkey[j][i];
			construc_truth_table_dynamic_CutM(cube, dim, roundnum,value0,rownum,columnum,key,k);

			for(i=0;i<10;i++)
				key[i]=randomkey[j+1][i];
			construc_truth_table_dynamic_CutM(cube, dim, roundnum,constantterm1,rownum,columnum,key,k);
			for(i=0;i<10;i++)
				key[i]=randomkey[j][i]^randomkey[j+1][i];
			construc_truth_table_dynamic_CutM(cube, dim, roundnum,constantterm1,rownum,columnum,key,k);
			for(a=0;a<rownum;a++){
				for(b=0;b<columnum;b++)
				{
					constantterm1[a][b] = constantterm[a][b]^constantterm1[a][b]^value0[a][b];
					value0[a][b]= constantterm[a][b]^value0[a][b];
				}
				Moebius(constantterm1[a], U64C(0x01)<<dim);
				Moebius(value0[a], U64C(0x01)<<dim);
			}
			//����Moebius�任֮��, ��Ҫ����ɸѡ, ��θ�Ч��ɸѡ, �����������,�������ж���Ȼ�ǲ���ȡ��,��Ȼ�ڵ�ά��ʱ����ô����
			//����ɸѡ��Ĵ����Filter[],�䳤��ΪFlength
			
			//m=(0x01<<dim)<<partdim;


			//m=(0x01<<(dim-partdim));
			//a=0;//���Կ�������
			//for(b=0;b<m;b++)
			//{
			//	weight=0;
			//	l=b;
			//	for(i=0;i<dim;i++)
			//	{
			//		weight+=(l&0x01);
			//		l>>=1;
			//	}
			//	if(weight>9)
			//	{
			//		Filter[Flength>>5]|=((constantterm1[a][b>>5]>>(b&0x01f))&0x01)<<(Flength&0x1f);
			//		Filter1[Flength>>5]|=((value0[a][b>>5]>>(b&0x01f))&0x01)<<(Flength&0x1f);
			//		//Fpos[Flength]=(b<<4)|k;
			//		Fpos[Flength]=(k<<12)|b;
			//		Flength++;
			//	}
			//}

		}
		//
		for(k=0;k<Flength;k++)
		{
			Fcheck[k]|=Filter[k];
			Fcheck1[k]|=Filter1[k];
		}
	
			for(b=0;b<n;b++)
			{
				Filter[b]=0;
				Filter1[b]=0;
			}
	}
	fprintf(fp2,"*************************\n");
	dim+=partdim;
	for(l=0;l<Flength;l++)
	{
		//b=Fpos[l];
		b=l;
		flag = Fcheck[b>>5] & ((U64C(0x01))<<(b&0x0000001F));
		flag1 = Fcheck1[b>>5] & ((U64C(0x01))<<(b&0x0000001F));
		//flag1=1;
		subdim=0;
		//if((flag==0))//ͨ�����Լ��,���ǲ�ͨ����ֵ����
		b=Fpos[l];
		if((flag==0)&&(flag1!=0))//ͨ�����Լ��,���ǲ�ͨ����ֵ����
		{
			for(i=0;i<dim;i++)
			{
				if(((b>>i)&0x01) ==1)
				{
					fprintf(fp2,"%d,",cube[i]);
					subdim++;
				}
			}
			fprintf(fp2,"\n");
			fprintf(Sdim,"%d,",subdim);
			outputsubcubes(cube,dim,roundnum,a,b,filename);
		}
	}
	
	for(i=0;i<rownum;i++)
	{
		free(constantterm[i]);
		free(constantterm1[i]);
		free(ANF[i]);
	}
	free(constantterm);
	free(constantterm1);
	free(ANF);
	free(check);
	fclose(fp2);
	fclose(Sdim);
	return cubenum;
}

//�̶�4��iv��Ԫ
u32 linearity_test_dynamicV4(u32 cube[], u32 dim, u32 roundnum,char filename[])
{
	u32 i,j,a,b,key[10],k,partdim,l,Flength;
	u32 **constantterm,**constantterm1,**check1,**value0;//������
	u32 numrandomkey = 16*2;  // ����һ��
	u32 randomkey[16*2][10];
	u32 **check,**ANF;//������Ҫ�ų�������
	FILE *fp2,*Sdim;
	u32 flag,flag1,subdim;
	u32 rownum = 1,cubenum=0;
	u32 columnum = U64C(0x01)<<(dim-5-4);
	fp2=fopen("CRec.txt","a+");
	Sdim=fopen("subdim.txt","a+");
	constantterm = (u32 **)malloc(rownum*sizeof(u32*));
	
	for(i=0;i<rownum;i++)
		constantterm[i]=(u32*)malloc(columnum*sizeof(u32));

	constantterm1 = (u32 **)malloc(rownum*sizeof(u32*));
	for(i=0;i<rownum;i++)
		constantterm1[i]=(u32*)malloc(columnum*sizeof(u32));
	
	check1 = (u32 **)malloc(rownum*sizeof(u32*));
	value0 = (u32 **)malloc(rownum*sizeof(u32*));
	ANF = (u32 **)malloc(rownum*sizeof(u32*));
	for(i=0;i<rownum;i++)
	{
		ANF[i]=(u32*)malloc(columnum*sizeof(u32));
		check1[i]=(u32*)malloc(columnum*sizeof(u32));
		value0[i]=(u32*)malloc(columnum*sizeof(u32));
	}

	check = (u32 **)malloc(rownum*sizeof(u32*));
	for(i=0;i<rownum;i++)
	{
		check[i]=(u32*)malloc(columnum*sizeof(u32));
	}

	for(a=0;a<rownum;a++)
		for(b=0;b<columnum;b++)
		{
				check[a][b] = 0;
				check1[a][b] = 0;
				constantterm[a][b]=0;
		}

	//�������һ��ѭ��
	//partdim=4;
	partdim=4;
	for(i=0;i<numrandomkey;i++)
		choose_random_key(randomkey[i]);
	//dim-=partdim;


	//������
	for(k=0;k<16;k++)
	{
		for(i=0;i<10;i++)
			key[i]= 0;
		construc_truth_table_dynamic_CutM(cube, dim, roundnum,constantterm,rownum,columnum,key,k);
	}
	for(j=0;j<numrandomkey;j=j+2)
	{
		for(a=0;a<rownum;a++)
			for(b=0;b<columnum;b++)
			{
					value0[a][b] = 0;
					constantterm1[a][b] = 0;
			}
		for(k=0;k<16;k++)
		{
			for(i=0;i<10;i++)
				key[i] = randomkey[j][i];
			construc_truth_table_dynamic_CutM(cube, dim, roundnum,value0,rownum,columnum,key,k);
		}
		for(k=0;k<16;k++)
		{
			for(i=0;i<10;i++)
				key[i]=randomkey[j+1][i];
			construc_truth_table_dynamic_CutM(cube, dim, roundnum,constantterm1,rownum,columnum,key,k);
		}
		for(k=0;k<16;k++)
		{
			for(i=0;i<10;i++)
				key[i]=randomkey[j][i]^randomkey[j+1][i];
			construc_truth_table_dynamic_CutM(cube, dim, roundnum,constantterm1,rownum,columnum,key,k);
		}
		for(a=0;a<rownum;a++)
		{
			for(b=0;b<columnum;b++)
			{
				constantterm1[a][b] = constantterm[a][b]^constantterm1[a][b]^value0[a][b];
				value0[a][b]= constantterm[a][b]^value0[a][b];
			}
			Moebius(constantterm1[a], U64C(0x01)<<(dim-partdim));
			Moebius(value0[a], U64C(0x01)<<(dim-partdim));;
			for(b=0;b<columnum;b++)
			{
				check[a][b] |= constantterm1[a][b];
				check1[a][b] |= value0[a][b];
			}
		}
			
		}
	fprintf(fp2,"*************************\n");
	
	Flength=(0x01)<<(dim-partdim);
	//dim+=partdim;
	a=0;
	for(l=0;l<Flength;l++)
	{
		b=l;
		flag = check[0][b>>5] & ((U64C(0x01))<<(b&0x0000001F));
		flag1 = check1[0][b>>5] & ((U64C(0x01))<<(b&0x0000001F));
		//flag1=1;
		subdim=0;
		b=0x0f<<(dim-partdim)|b;
		//if((flag==0))//ͨ�����Լ��,���ǲ�ͨ����ֵ����
		if((flag==0)&&(flag1!=0))//ͨ�����Լ��,���ǲ�ͨ����ֵ����
		{
			for(i=0;i<dim;i++)
			{
				if(((b>>i)&0x01) ==1)
				{
					fprintf(fp2,"%d,",cube[i]);
					subdim++;
				}
			}
			fprintf(fp2,"\n");
			fprintf(Sdim,"%d,",subdim);
			outputsubcubes(cube,dim,roundnum,a,b,filename);
		}
	}
	
	for(i=0;i<rownum;i++)
	{
		free(constantterm[i]);
		free(constantterm1[i]);
		free(ANF[i]);
	}
	free(constantterm);
	free(constantterm1);
	free(ANF);
	free(check);
	fclose(fp2);
	fclose(Sdim);
	return cubenum;
}


////��ɸѡ, Ȼ����һ��16Ԫ��Meobius�任

u32 linearity_test_dynamicV5(u32 cube[], u32 dim, u32 roundnum,char filename[])
{
	u32 i,j,a,b,key[10],k,partdim=4,weight,l,m,n,Sz,Pos;
	u32 **constantterm,**constantterm1,**value0;//������
	u32 **Filter1,**Filter, Flength=156227, **Fcheck,**Fcheck1,**Fpos;
	u32 numrandomkey = 16*2;  // ����һ��
	u32 randomkey[16*2][10];
	FILE *fp2,*Sdim;
	u32 flag,flag1,subdim;
	u32 rownum = 1,cubenum=0;
	u32 columnum = U64C(0x01)<<(dim-5-partdim);
	clock_t t1,t2;
	fp2=fopen("CRec.txt","a+");
	Sdim=fopen("subdim.txt","a+");
	n=Flength;
	constantterm = (u32 **)malloc(rownum*sizeof(u32*));
	Filter1=(u32 **) malloc(sizeof(u32*)*Flength);
	Filter=(u32 **) malloc(sizeof(u32*)*Flength);//2ά����,���ڱ���ÿһ�����ֺ�����ANF(ɸѡ��)
	Fcheck=(u32 **) malloc(sizeof(u32*)*Flength);
	Fcheck1=(u32 **) malloc(sizeof(u32*)*Flength);
	Fpos=(u32 **) malloc(sizeof(u32*)*Flength);
	for(i=0;i<rownum;i++)
		constantterm[i]=(u32*)malloc(columnum*sizeof(u32));

	for(i=0;i<16;i++)
	{
		Filter1[i]=(u32*)malloc(Flength*sizeof(u32));
		Filter[i]=(u32*)malloc(Flength*sizeof(u32));
		Fpos[i]=(u32*)malloc(Flength*sizeof(u32));
		Fcheck[i]=(u32*)malloc(Flength*sizeof(u32));
		Fcheck1[i]=(u32*)malloc(Flength*sizeof(u32));
	}
	constantterm1 = (u32 **)malloc(rownum*sizeof(u32*));
	for(i=0;i<rownum;i++)
		constantterm1[i]=(u32*)malloc(columnum*sizeof(u32));
	
	value0 = (u32 **)malloc(rownum*sizeof(u32*));
	for(i=0;i<rownum;i++)
	{
		value0[i]=(u32*)malloc(columnum*sizeof(u32));
	}


	//�������һ��ѭ��
	//partdim=4;
	partdim=4;
	for(i=0;i<numrandomkey;i++)
		choose_random_key(randomkey[i]);
	//dim-=partdim;
	for(a=0;a<16;a++)
		for(b=0;b<Flength;b++)
		{
			Filter[a][b]=0;
			Filter1[a][b]=0;
			Fpos[a][b]=0;
			Fcheck[a][b]=0;
			Fcheck1[a][b]=0;
		}

		//��ɸѡ����
		m=(0x01<<(dim-partdim));
		Flength=0;
		for(b=0;b<m;b++)
			{
				weight=0;
				l=b;
				for(i=0;i<dim;i++)
				{
					weight+=(l&0x01);
					l>>=1;
				}
				if(weight>dim-7)
				{
				//	Filter[k][Flength>>5]|=((constantterm1[a][b>>5]>>(b&0x01f))&0x01)<<(Flength&0x1f);
				//	Filter1[k][Flength>>5]|=((value0[a][b>>5]>>(b&0x01f))&0x01)<<(Flength&0x1f);
					//Fpos[Flength]=(b<<4)|k;
					Fpos[0][Flength]=(0<<(dim-partdim))|b;
					Flength++;
				}
			}
	
		//dim=dim+4;
	//t1=clock();
	for(j=0;j<numrandomkey;j=j+2)
	{
		//Flength=0;
		t1=clock();
		for(k=0;k<16;k++)
		{
		//	Flength=0;
		
			for(a=0;a<rownum;a++)
				for(b=0;b<columnum;b++)
				{
						constantterm[a][b] = 0;
						value0[a][b] = 0;
						constantterm1[a][b] = 0;
				}
			//ȡcube,Ȼ����㳣ֵ
		for(i=0;i<10;i++)
			key[i]= 0;
		construc_truth_table_dynamic_CutM_32(cube, dim, roundnum,constantterm,rownum,columnum,key,k);
		//construc_truth_table_dynamicV2(cube, dim-6, roundnum,constantterm,rownum,columnum,key);
		for(i=0;i<10;i++)
			key[i] = randomkey[j][i];
		construc_truth_table_dynamic_CutM_32(cube, dim, roundnum,value0,rownum,columnum,key,k);

			for(i=0;i<10;i++)
				key[i]=randomkey[j+1][i];
		construc_truth_table_dynamic_CutM_32(cube, dim, roundnum,constantterm1,rownum,columnum,key,k);
			for(i=0;i<10;i++)
				key[i]=randomkey[j][i]^randomkey[j+1][i];
    	construc_truth_table_dynamic_CutM_32(cube, dim, roundnum,constantterm1,rownum,columnum,key,k);
			for(a=0;a<rownum;a++){
				for(b=0;b<columnum;b++)
				{
					constantterm1[a][b] = constantterm[a][b]^constantterm1[a][b]^value0[a][b];
					value0[a][b]= constantterm[a][b]^value0[a][b];
				}
				Moebius(constantterm1[a], U64C(0x01)<<(dim-partdim));
				Moebius(value0[a], U64C(0x01)<<(dim-partdim));
			}
			a=0;
			for(b=0;b<Flength;b++)
			{
					//Fpos[k][b]=(k<<(dim-partdim))|((Fpos[0][b]&()partdim)>>partdim);
				//��4λ����0��ֱ�ӻ��ϾͿ���
				Fpos[k][b]=(k<<(dim-partdim))|(Fpos[0][b]);
				Filter[k][b>>5]|=((constantterm1[a][Fpos[0][b]>>5]>>(Fpos[0][b]&0x01f))&0x01)<<(b&0x1f);
				Filter1[k][b>>5]|=((value0[a][Fpos[0][b]>>5]>>(Fpos[0][b]&0x01f))&0x01)<<(b&0x1f);
					//Fpos[Flength]=(b<<4)|k;
					//(k<<(dim-partdim))|b;
				//	Flength++;
			}

			//m=(0x01<<(dim-partdim));
			//a=0;//���Կ�������
			//for(b=0;b<m;b++)
			//{
			//	weight=0;
			//	l=b;
			//	for(i=0;i<dim;i++)
			//	{
			//		weight+=(l&0x01);
			//		l>>=1;
			//	}
			//	if(weight>dim-7)
			//	{
			//		Filter[k][Flength>>5]|=((constantterm1[a][b>>5]>>(b&0x01f))&0x01)<<(Flength&0x1f);
			//		Filter1[k][Flength>>5]|=((value0[a][b>>5]>>(b&0x01f))&0x01)<<(Flength&0x1f);
			//		//Fpos[Flength]=(b<<4)|k;
			//		Fpos[k][Flength]=(k<<(dim-partdim))|b;
			//		Flength++;
			//	}
			//}
		
		}
		t2=clock();
		printf("%dms\n",t2-t1);
		//����, ������洢�ĸ���������ֵ������ֶ�Moebius�任
		k=(0x01<<partdim);
		for(i=0;i<partdim;i++)
		{
			Sz=0x01<<i;
			Pos=0;
			while(Pos<k)
			{
				for(b=0;b<Sz;b++)
				{
					for(a=0;a<Flength;a++)
					{
						Filter[Pos+Sz+b][a]=Filter[Pos+Sz+b][a]^Filter[Pos+b][a];
						Filter1[Pos+Sz+b][a]=Filter1[Pos+Sz+b][a]^Filter1[Pos+b][a];
					}

				}
				Pos=Pos+2*Sz;
			}
		}
		for(a=0;a< U64C(0x01)<<(partdim);a++)
		{
			for(k=0;k<Flength;k++)
			{
				Fcheck[a][k]|=Filter[a][k];
				Fcheck1[a][k]|=Filter1[a][k];
			}
		}
		k=0x01<<partdim;
		for(a=0;a<k;a++)
		{
			for(b=0;b<n;b++)
			{
				Filter[a][b]=0;
				Filter1[a][b]=0;
			}
		}
	}

	fprintf(fp2,"*************************\n");
	//dim+=partdim;
	t1=clock();
	n=(0x01<<partdim);
	a=0;
	for(i=0;i<n;i++)
	{
		for(l=0;l<Flength;l++)
		{
			//b=Fpos[l];
			b=l;
			flag = Fcheck[i][b>>5] & ((U64C(0x01))<<(b&0x0000001F));
			flag1 = Fcheck1[i][b>>5] & ((U64C(0x01))<<(b&0x0000001F));
			//flag1=1;
			subdim=0;
			//if((flag==0))//ͨ�����Լ��,���ǲ�ͨ����ֵ����
			b=Fpos[i][l];
			if((flag==0)&&(flag1!=0))//ͨ�����Լ��,���ǲ�ͨ����ֵ����
			{
				for(j=0;j<dim;j++)
				{
					if(((b>>j)&0x01) ==1)
					{
						fprintf(fp2,"%d,",cube[j]);
						subdim++;
					}
				}
				fprintf(fp2,"\n");
				fprintf(Sdim,"%d,",subdim);
				outputsubcubes(cube,dim,roundnum,a,b,filename);
			}
		}
	}
	/*t2=clock();
	printf("%dms\n",t2-t1);*/
	for(i=0;i<rownum;i++)
	{
		free(constantterm[i]);
		free(constantterm1[i]);
		free(value0[i]);
	}
	free(constantterm);
	free(constantterm1);
	fclose(fp2);
	fclose(Sdim);
		for(i=0;i<16;i++)
	{
		free(Filter[i]);
		free(Filter1[i]);
		free(Fcheck[i]);
		free(Fcheck1[i]);
		free(Fpos[i]);
	}
	free(Filter);
	free(Filter1);
	free(Fcheck);
	free(Fcheck1);
	free(Fpos);
	return cubenum;
}



/////��϶��ε�������ʽ�������Լ�����߶�����
///K1��K2�������ѡ�� ֻҪ������ʽ��K1+K2����4�Σ�ά�������ӵ�ʱ����Ҫ���ã����Ǹ�����Ҫ����Ĵ洢�ռ䣬ֻ��Ҫ�洢ɸѡ����� 

//���������ѡȡ������Կ��Ȼ��ֱ���������ֻ�


///������ʽ���Լ��
__host__ u8 specialform_test_randomkey_word(u32 cube[], u32 dim, u32 roundnum,u32 randomkey[][10],u32 twokey[][10],u8 linearity_test_res[])
{
	u32 chosenkey[48+1][10],i,keynum,partialsum[1024]={0},value[48]={0},j;
	//���鳤�Ȼ��д���ȶ,��Ҫ����ά������ȷ��
	//��Ҫ���ݵĲ���,�����Կ,cube,roundnum,����Ҫ������һ��ռ����ڴ�ŷ��صĲ������ֵ,�������Ͻ������,����һ������
	u32 **chosenkey_dev,*partialsum_dev;
	u32 blocksPerGrid= 64;//(0x01<<dim)/threadsPerBlock;
	u32 constantterm=0;
	u32 length,check=0,check_con=0;
	u32 *cube_dev,res,check_bit,flag=0;
	//size_t pitch;
	keynum=48;
	length=blocksPerGrid;//((0x01<<dim)>>4)>>8;

	hipMalloc((void **)&cube_dev,dim*sizeof(u32));
	hipMalloc((void **)&partialsum_dev,length*sizeof(u32));
	chosenkey_dev=(u32 **)malloc(sizeof(u32 *)*(keynum+1));
	for(i=0;i<keynum+1;i++)
	{
		hipMalloc((void **) &chosenkey_dev[i],10*sizeof(u32));
		
	}

	for(i=1;i<keynum+1;i+=3)
	{
		/////������Կ
		for(j=0;j<10;j++)
		{
			chosenkey[i][j]=randomkey[(i/3)*2][j];
			chosenkey[i+1][j]=randomkey[(i/3)*2+1][j];//
			chosenkey[i+2][j]=twokey[i/3][j];
		}
	}

	for(i=0;i<10;i++)
		chosenkey[0][i]=0;
	//�������˵����ݿ������豸��
	hipMemcpy(cube_dev,cube,dim*sizeof(u32),hipMemcpyHostToDevice);
	for(i=0;i<keynum+1;i++)
	{
		hipMemcpy(chosenkey_dev[i],chosenkey[i],sizeof(u32)*10,hipMemcpyHostToDevice);
	}
	hipEvent_t start,stop;
	hipEventCreate(&start);  
    hipEventCreate(&stop);  
    hipEventRecord(start,0);  
	sum_cube_word<<<blocksPerGrid,threadsPerBlock>>>(dim,cube_dev,roundnum,partialsum_dev,chosenkey_dev[0]);
	hipMemcpy(partialsum,partialsum_dev,sizeof(u32)*length,hipMemcpyDeviceToHost);
	hipEventRecord(stop,0);  
    hipEventSynchronize(stop);
	float tm;  
    hipEventElapsedTime(&tm,start,stop);  
    printf("GPU Elapsed time:%.6f ms.\n",tm);  
	for(i=0;i<length;i++)
		constantterm^=partialsum[i];
	for(j=0;j<keynum;j+=3)
	{
		//key1
		sum_cube_word<<<blocksPerGrid,threadsPerBlock>>>(dim,cube_dev,roundnum,partialsum_dev,chosenkey_dev[j+1]);
		hipMemcpy(partialsum,partialsum_dev,sizeof(u32)*length,hipMemcpyDeviceToHost);
		for(i=0;i<length;i++)
			value[j]^=partialsum[i];
		//key2
		sum_cube_word<<<blocksPerGrid,threadsPerBlock>>>(dim,cube_dev,roundnum,partialsum_dev,chosenkey_dev[j+2]);
		hipMemcpy(partialsum,partialsum_dev,sizeof(u32)*length,hipMemcpyDeviceToHost);
		for(i=0;i<length;i++)
			value[j+1]^=partialsum[i];
		//key1+key2
		sum_cube_word<<<blocksPerGrid,threadsPerBlock>>>(dim,cube_dev,roundnum,partialsum_dev,chosenkey_dev[j+3]);
		hipMemcpy(partialsum,partialsum_dev,sizeof(u32)*length,hipMemcpyDeviceToHost);
		for(i=0;i<length;i++)
			value[j+2]^=partialsum[i];
		//
		check_con = check_con| (constantterm^value[j]);
		check = check | (constantterm^value[j]^value[j+1]^value[j+2]);
		if(check == 0xFFFFFFFF)
			break;
	}
	if(check==0xFFFFFFFF)//Ҫ�ĳ�FFFFFFFF����������
	{
		for(i=0;i<32;i++)
			linearity_test_res[i] = 0;
		/*for(i=0;i<32;i++)
			printf("%d,",linearity_test_res[i]);
		printf("\n");*/
		return 0;
	}
	else{
		res = 2;
		for(i=0;i<32;i++)
		{
			check_bit = (check>>i) & 0x01;
			if(check_bit==0x01)
				linearity_test_res[i]=0;
			else
			{
				flag = 0;
				//����Ƿ�ֵ
				if((check_con>>i) & 0x01)
					linearity_test_res[i] = 1;
				else
					linearity_test_res[i] = 2;
			}	
		}
	}

	hipFree(cube_dev);
	hipFree(partialsum_dev);
	for(i=0;i<keynum+1;i++)
		hipFree(chosenkey_dev[i]);
	//cudaEvent
	hipEventDestroy(start);
	hipEventDestroy(stop);
	return res;
}
// ������ʽ�Ĳ�ֵ

__host__ void write_superpoly_subcubeSF(FILE * fp,u32 cube[], u8 dim,u32 roundnum, u8 outindex)
{
	u32 chosenkey[80+1][10],i,keynum,partialsum[1024]={0},value[81]={0},j;
	//���鳤�Ȼ��д���ȶ,��Ҫ����ά������ȷ��
	//��Ҫ���ݵĲ���,�����Կ,cube,roundnum,����Ҫ������һ��ռ����ڴ�ŷ��صĲ������ֵ,�������Ͻ������,����һ������
	u32 **randkey_dev,*psum_dev,**randkey_dev2;
	u32 blocksPerGrid= 64;//(0x01<<dim)/threadsPerBlock;
	u32 constantterm=0, equnum=80,testnum=32;
	u32 length;
	u32 *cube_d,flag[80]={0},flag1,flag2,flag3,flag4;
	u8 coeff[81]={0},temp2,temp,k;
	//size_t pitch;
	keynum=80;

	length=blocksPerGrid;//((0x01<<dim)>>4)>>8;
	///length=((0x01<<dim);
	hipMalloc((void **)&cube_d,dim*sizeof(u32));
	hipMalloc((void **)&psum_dev,length*sizeof(u32));
	randkey_dev=(u32 **)malloc(sizeof(u32 *)*(keynum+1));
	randkey_dev2=(u32 **)malloc(sizeof(u32 *)*(testnum));
	for(i=0;i<keynum+1;i++)
	{
		hipMalloc((void **) &randkey_dev[i],10*sizeof(u32));
		
	}
	for(i=0;i<testnum;i++)
	{
		hipMalloc((void **) &randkey_dev2[i],10*sizeof(u32));	
	}
	for(i=1;i<keynum+1;i++)
	{
		for(j=0;j<10;j++)
			chosenkey[i][j]=0;
		//chosenkey[i][(i-1)>>3]=(0x01)<<((i-1)&0x07);
	}
	for(i=0;i<10;i++)
		chosenkey[0][i]=0;
	//�������˵����ݿ������豸��
	hipMemcpy(cube_d,cube,dim*sizeof(u32),hipMemcpyHostToDevice);
	///������Կ���豸��


	for(i=0;i<equnum;i++)
	{
		hipMemcpy(randkey_dev[i],chosenkey[i],sizeof(u32)*10,hipMemcpyHostToDevice);
	}
	//����2�β�ֵ��Ȼ�������Բ�ֵ

	for(j=0;j<equnum;j++)
	{
		//printf("START>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>\n");
		flag1=0;flag2=0;flag3=0;flag4=0;
		sum_cube_word_sf<<<blocksPerGrid,threadsPerBlock>>>(dim,cube_d,roundnum,psum_dev,randkey_dev[j],j,0);
		hipMemcpy(partialsum,psum_dev,sizeof(u32)*length,hipMemcpyDeviceToHost);
		for(i=0;i<length;i++)
			flag1^=partialsum[i];
		flag1=(flag1>>outindex)&0x01;

		sum_cube_word_sf<<<blocksPerGrid,threadsPerBlock>>>(dim,cube_d,roundnum,psum_dev,randkey_dev[j],j,1);
		hipMemcpy(partialsum,psum_dev,sizeof(u32)*length,hipMemcpyDeviceToHost);
		for(i=0;i<length;i++)
			flag2^=partialsum[i];
		flag2=(flag2>>outindex)&0x01;

		sum_cube_word_sf<<<blocksPerGrid,threadsPerBlock>>>(dim,cube_d,roundnum,psum_dev,randkey_dev[j],j,2);
		hipMemcpy(partialsum,psum_dev,sizeof(u32)*length,hipMemcpyDeviceToHost);
		for(i=0;i<length;i++)
			flag3^=partialsum[i];
		flag3=(flag3>>outindex)&0x01;
		
		sum_cube_word_sf<<<blocksPerGrid,threadsPerBlock>>>(dim,cube_d,roundnum,psum_dev,randkey_dev[j],j,3);
		hipMemcpy(partialsum,psum_dev,sizeof(u32)*length,hipMemcpyDeviceToHost);
		for(i=0;i<length;i++)
			flag4^=partialsum[i];
		flag4=(flag4>>outindex)&0x01;

		temp=((flag1^flag2^flag3^flag4)>>outindex)&0x01;
	//	printf("DONE>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>\n");
		if(temp==1)
		{
			//�����Կ��Ȼ�󿽱����豸�ˣ� Ȼ��
			for(k=0;k<testnum;k++)
				choose_random_key(chosenkey[k]);
			for(k=0;k<testnum;k++)
				hipMemcpy(randkey_dev2[k],chosenkey[k],sizeof(u32)*10,hipMemcpyHostToDevice);

			for(k=0;k<testnum;k++)
			{
				flag1=0;flag2=0;flag3=0;flag4=0;
				sum_cube_word_sf<<<blocksPerGrid,threadsPerBlock>>>(dim,cube_d,roundnum,psum_dev,randkey_dev2[k],j,0);
				hipMemcpy(partialsum,psum_dev,sizeof(u32)*length,hipMemcpyDeviceToHost);
				for(i=0;i<length;i++)
					flag1^=partialsum[i];
				flag1=(flag1>>outindex)&0x01;

				sum_cube_word_sf<<<blocksPerGrid,threadsPerBlock>>>(dim,cube_d,roundnum,psum_dev,randkey_dev2[k],j,1);
				hipMemcpy(partialsum,psum_dev,sizeof(u32)*length,hipMemcpyDeviceToHost);
				for(i=0;i<length;i++)
					flag2^=partialsum[i];
				flag2=(flag2>>outindex)&0x01;

				sum_cube_word_sf<<<blocksPerGrid,threadsPerBlock>>>(dim,cube_d,roundnum,psum_dev,randkey_dev2[k],j,2);
				hipMemcpy(partialsum,psum_dev,sizeof(u32)*length,hipMemcpyDeviceToHost);
				for(i=0;i<length;i++)
					flag3^=partialsum[i];
				flag3=(flag3>>outindex)&0x01;

				sum_cube_word_sf<<<blocksPerGrid,threadsPerBlock>>>(dim,cube_d,roundnum,psum_dev,randkey_dev2[k],j,3);
				hipMemcpy(partialsum,psum_dev,sizeof(u32)*length,hipMemcpyDeviceToHost);
				for(i=0;i<length;i++)
					flag4^=partialsum[i];
				flag4=(flag4>>outindex)&0x01;

				temp2=((flag1^flag2^flag3^flag4)>>outindex)&0x01;
				if(temp2!=temp)
				{
					break;
				}
			}
			if(k==testnum)//�����k=keynum,˵����ô��εļ�ⶼ��ͨ����,
			{
				fprintf(fp,"x%d*x%d+",j,j+1);
				flag[j]=1;
				flag[j+1]=1;
				//flag[i+2]=1;
				//sig=0;
			}
		}
		
	}

	//���Բ���Ҫ��ֱ��ʡ��
	for(i=1;i<keynum+1;i++)
	{
		for(j=0;j<10;j++)
			chosenkey[i][j]=0;
		chosenkey[i][(i-1)>>3]=(0x01)<<((i-1)&0x07);
	}

	for(i=0;i<10;i++)
		chosenkey[0][i]=0;

	for(i=0;i<keynum+1;i++)
	{
		hipMemcpy(randkey_dev[i],chosenkey[i],sizeof(u32)*10,hipMemcpyHostToDevice);
	}

	hipEvent_t start,stop;
	hipEventCreate(&start);  
    hipEventCreate(&stop);  
    hipEventRecord(start,0);  
	sum_cube_word<<<blocksPerGrid,threadsPerBlock>>>(dim,cube_d,roundnum,psum_dev,randkey_dev[0]);
	//sum_cube_wordV2<<<blocksPerGrid,threadsPerBlock>>>(dim,cube_d,roundnum,psum_dev,0);
	hipEventRecord(stop,0);  
    hipEventSynchronize(stop);  
    float tm;  
    hipEventElapsedTime(&tm,start,stop);  
    printf("GPU Elapsed time:%.6f ms.\n",tm);  

	hipMemcpy(partialsum,psum_dev,sizeof(u32)*length,hipMemcpyDeviceToHost);
	for(i=0;i<length;i++)
		constantterm^=partialsum[i];
	coeff[0]=(constantterm>>outindex)&(0x01);
	
	
	for(j=1;j<keynum+1;j++) 
	{
		if(flag[j]==0)
		{
		sum_cube_word<<<blocksPerGrid,threadsPerBlock>>>(dim,cube_d,roundnum,psum_dev,randkey_dev[j]);
		hipMemcpy(partialsum,psum_dev,sizeof(u32)*length,hipMemcpyDeviceToHost);
		for(i=0;i<length;i++)
			value[j]^=partialsum[i];
		coeff[j]=((value[j]>>outindex)&0x01)^coeff[0];
		if(coeff[j]==1)
			fprintf(fp,"x%d+",j-1);
		}	
	}
	fprintf(fp,"%d",coeff[0]);
	 fprintf(fp, "\n");
	//u32 **randkey_dev,*psum_dev;
	 hipFree(cube_d);
	hipFree(psum_dev);
	for(i=0;i<keynum+1;i++)
		hipFree(randkey_dev[i]);
	free(randkey_dev);
	hipEventDestroy(start);
	hipEventDestroy(stop);
}

//������ʽ�����
//void outputcubeofsf(u32 cube[], u8 dim, u32 roundnum, u8 outindex, u32 pos, char filename[],u32 randomkey[][10],u32 twokey[][10])
	void outputcubeofsf(u32 cube[], u8 dim, u32 roundnum, u8 outindex, unsigned __int64 pos, char filename[],u32 randomkey[][10],u32 twokey[][10])
{
	u8 i,j;
	u32 *subcube;
	u8 subdim;
	u8 result=0x00;
	u8 linearity_test_res[32];
	FILE * fp;

	subcube = (u32 *)malloc(dim*sizeof(u32));

	fp = fopen(filename,"a+");
	subdim = 0;
	for (j=0;j<dim;j++) {
		if((pos>>j) & 0x00000001)
		{
			subcube[subdim] = cube[j];
			subdim++;
		}
	}	

	//flag=linearity_test_randomkey_word(subcube, subdim, roundnum,linearity_test_res);
	result=specialform_test_randomkey_word(subcube, subdim, roundnum,randomkey,twokey,linearity_test_res);
	if((linearity_test_res[outindex]==1))
	{
		if(firstoutput)
		{
			fprintf(fp, "\n\nround = %d, dim = %d,",roundnum,dim);
			fprintf(fp, "\nmother cube: ");
			for(i=0;i<dim;i++)
				fprintf(fp, "%d,",cube[i]);
			firstoutput = 0;
		}
		fprintf(fp, "\nround = %d, subdim = %d", roundnum+outindex,subdim);
		fprintf(fp,"\nTYPE2: subcube: ");
		for(j=0;j<subdim;j++)
			fprintf(fp,"%d,",subcube[j]);
		fprintf(fp,"	");
		//write_superpoly_subcube(fp,subcube, subdim,roundnum, outindex);
		//write_superpoly_sfcube(fp,subcube, subdim,roundnum, outindex);
		//write_superpoly_sfcubeV2(fp,subcube, subdim,roundnum, outindex);
		write_superpoly_subcubeSF(fp,subcube, subdim,roundnum, outindex);
	}


	fclose(fp);
	free(subcube);
}

void outputcubeofsf2(u32 cube[], u8 dim, u32 roundnum, u8 outindex, unsigned __int64 pos, char filename[],u32 randomkey[][10],u32 twokey[][10])
{
	u8 i,j;
	u32 *subcube;
	u8 subdim;
	u8 result=0x00;
	u8 linearity_test_res[32];
	FILE * fp;

	subcube = (u32 *)malloc(dim*sizeof(u32));

	fp = fopen(filename,"a+");
	subdim = 0;
	for (j=0;j<dim;j++) {
		if((pos>>j) & 0x00000001)
		{
			subcube[subdim] = cube[j];
			subdim++;
		}
	}	

	//flag=linearity_test_randomkey_word(subcube, subdim, roundnum,linearity_test_res);
	result=specialform_test_randomkey_word(subcube, subdim, roundnum,randomkey,twokey,linearity_test_res);
	if((linearity_test_res[outindex]==1))
	{
		if(firstoutput)
		{
			fprintf(fp, "\n\nround = %d, dim = %d,",roundnum,dim);
			fprintf(fp, "\nmother cube: ");
			for(i=0;i<dim;i++)
				fprintf(fp, "%d,",cube[i]);
			firstoutput = 0;
		}
		fprintf(fp, "\nround = %d, subdim = %d", roundnum+outindex,subdim);
		fprintf(fp,"\nTYPE3: subcube: ");
		for(j=0;j<subdim;j++)
			fprintf(fp,"%d,",subcube[j]);
		fprintf(fp,"	");
		//write_superpoly_subcube(fp,subcube, subdim,roundnum, outindex);
		//write_superpoly_sfcube(fp,subcube, subdim,roundnum, outindex);
		//write_superpoly_sfcubeV2(fp,subcube, subdim,roundnum, outindex);
		write_superpoly_subcubeSF(fp,subcube, subdim,roundnum, outindex);
	}


	fclose(fp);
	free(subcube);
}
//��Ҫע�����Flength��ȡֵ�Լ�dim�Ľ�ȡ
u32 linearity_test_dynamicV6(u32 cube[], u32 dim, u32 roundnum,int **canindex, int cannum, char filename[])
{
	unsigned __int64  i,j,a,b,weight,l,m,n,Sz,Pos,tt=0,temp_M=0,k=0;
	u32 key[10],partdim=7;
	u32 **constantterm, **value0, **value1, **value2,**value3,**value4;//������2427000
	//u32 **Filter1,**Filter, Flength=2827000, **Fcheck,**Fcheck1, **sFilter1,**sFilter,**sFcheck,**sFcheck1;//3492176
	u32 **Filter1,**Filter, Flength=2427000, **Fcheck,**Fcheck1, **sFilter1,**sFilter,**sFcheck,**sFcheck1,**T2Filter1,**T2Filter,**T2Fcheck,**T2Fcheck1;//3492176
	unsigned __int64 	*Fpos;//���ڴ洢λ��
	u32 numrandomkey = 160,tttt;  // ����һ��
	u32 randomkey[256*2][10],sfkey[256*2][10]={0},sfkey2[256][10]={0},tempkey[256][10]={0};//��һ����Կ
	u32 randomkeypro[256*2][10]={0},sfkeypro[256*2][10]={0},sfkeypro2[256][10]={0};//�ڶ�����Կ
	int keynum=81;
	u8 breakflag=0;
	clock_t t1,t2;
	//u32 **ANF;//������Ҫ�ų�������
	FILE *fp2,*Sdim;
	partdim=7;
	u32 flag,flag1,subdim,flag2,flag3,flag4,flag5;
	u32 rownum = 1,cubenum=0;
	unsigned __int64  total, total1,total2,total3;
	u32 columnum = U64C(0x01)<<(dim-5-partdim);
	u32 **coeffs;
	int t=0;
	fp2=fopen("CRec.txt","a+");
	Sdim=fopen("subdim.txt","a+");
	n=Flength;
	coeffs=(u32 **)malloc(cannum*sizeof(u32*));
	constantterm = (u32 **)malloc(rownum*sizeof(u32*));
	Filter1=(u32 **) malloc(sizeof(u32*)*128);
	Filter=(u32 **) malloc(sizeof(u32*)*128);//2ά����,���ڱ���ÿһ�����ֺ�����ANF(ɸѡ��)
	Fcheck=(u32 **) malloc(sizeof(u32*)*128);
	Fcheck1=(u32 **) malloc(sizeof(u32*)*128);
	//Fpos=(unsigned __int64 **) malloc(sizeof(unsigned __int64  *)*Flength);
	Fpos=(unsigned __int64 *) malloc(sizeof(unsigned __int64 )*(Flength*32));//���ڱ���ɸѡ�����ĵ������, ����Ҫ����32

	sFilter1=(u32 **) malloc(sizeof(u32*)*128);//����128���ɣ���
	sFilter=(u32 **) malloc(sizeof(u32*)*128);//2ά����,���ڱ���ÿһ�����ֺ�����ANF(ɸѡ��)
	sFcheck=(u32 **) malloc(sizeof(u32*)*128);
	sFcheck1=(u32 **) malloc(sizeof(u32*)*128);

	T2Filter1=(u32 **) malloc(sizeof(u32*)*128);
	T2Filter=(u32 **) malloc(sizeof(u32*)*128);//2ά����,���ڱ���ÿһ�����ֺ�����ANF(ɸѡ��)
	T2Fcheck=(u32 **) malloc(sizeof(u32*)*128);
	T2Fcheck1=(u32 **) malloc(sizeof(u32*)*128);
	for(i=0;i<rownum;i++)
		constantterm[i]=(u32*)malloc(columnum*sizeof(u32));

	for(i=0;i<cannum;i++)
		coeffs[i]=(u32 *)malloc(sizeof(u32)*81);

	for(i=0;i<128;i++)
	{
		Filter1[i]=(u32*)malloc(Flength*sizeof(u32));
		Filter[i]=(u32*)malloc(Flength*sizeof(u32));
		//Fpos[i]=(unsigned __int64 *)malloc(Flength*sizeof(unsigned __int64));//���ȣ�Fposֻ��Ҫ�洢һ�Σ�
		Fcheck[i]=(u32*)malloc(Flength*sizeof(u32));
		Fcheck1[i]=(u32*)malloc(Flength*sizeof(u32));

		sFilter1[i]=(u32*)malloc(Flength*sizeof(u32));
		sFilter[i]=(u32*)malloc(Flength*sizeof(u32));
		sFcheck[i]=(u32*)malloc(Flength*sizeof(u32));
		sFcheck1[i]=(u32*)malloc(Flength*sizeof(u32));

		T2Filter1[i]=(u32*)malloc(Flength*sizeof(u32));
		T2Filter[i]=(u32*)malloc(Flength*sizeof(u32));
		T2Fcheck[i]=(u32*)malloc(Flength*sizeof(u32));
		T2Fcheck1[i]=(u32*)malloc(Flength*sizeof(u32));
	}
	
	value0 = (u32 **)malloc(rownum*sizeof(u32*));
	value1 = (u32 **)malloc(rownum*sizeof(u32*));
	value2 = (u32 **)malloc(rownum*sizeof(u32*));
	value3 = (u32 **)malloc(rownum*sizeof(u32*));
	value4 = (u32 **)malloc(rownum*sizeof(u32*));
	for(i=0;i<rownum;i++)
	{
		value0[i]=(u32*)malloc(columnum*sizeof(u32));
		value1[i]=(u32*)malloc(columnum*sizeof(u32));
		value2[i]=(u32*)malloc(columnum*sizeof(u32));
		value3[i]=(u32*)malloc(columnum*sizeof(u32));
		value4[i]=(u32*)malloc(columnum*sizeof(u32));
	}


	//�������һ��ѭ��
	partdim=7;
	//partdim=7;
	
	//����һ�������Կ
	srand((unsigned int) time(NULL));
	//for(i=0;i<numrandomkey;i++)
		//choose_random_key(randomkey[i]);//GenRandomKeyV3Right(randomkey,sfkey2);
	//genrandomkeyV2(randomkey,sfkey);//��������һ�������Կ, ̫��,00.06

	for(i=1;i<keynum;i++)
	{
		for(j=0;j<10;j++)
			randomkey[i][j]=0;
		randomkey[i][(i-1)>>3]=(0x01)<<((i-1)&0x07);
	}
	
	for(a=0;a<128;a++)
		for(b=0;b<Flength;b++)
		{
			Filter[a][b]=0;
			Filter1[a][b]=0;
			Fpos[b]=0;
			Fcheck[a][b]=0;
			Fcheck1[a][b]=0;

			sFilter[a][b]=0;
			sFilter1[a][b]=0;
			sFcheck[a][b]=0;
			sFcheck1[a][b]=0;

			T2Filter[a][b]=0;
			T2Filter1[a][b]=0;
			T2Fcheck[a][b]=0;
			T2Fcheck1[a][b]=0;
		}
	//������Ҫ�����õ����������������ʱ����Ҫ
	Flength=0;
	m=U64C(0x01)<<(dim-partdim);
	printf("Start get index>>>>\n");
	for(b=0;b<m;b++)
	{
		weight=0;
		l=b;
		for(i=0;i<dim;i++)
		{
			weight+=(l&0x01);
			l>>=1;
		}
		if(weight>(dim-partdim-9))//41-7-10+1=25+7=32//cong32kaishi
		//if(weight>21)//42-9+1//ֱ��д�����֣�Ȼ������֤һ���Ƿ���ȷ
		{
			Fpos[Flength]= (U64C(0x00)<<(dim-partdim))|b;
			Flength++;
		}
	}
		//�ܹ���ҪFlength���㣬ÿ������һ���Ӻ�������ֵ���Ժ�ɸȡ��Ӧλ�õ�0��1���뵽Filter�Ȼ���Filter����Moebius�任
	printf("Done>>>>\nStart  Computing Cube>>>>\n");
	printf("%u\n",Flength);
	printf("%llu\n",Fpos[Flength-1]);
	printf("%llu\n",Fpos[Flength-2]);
	printf("%llu\n",Fpos[Flength-3]);
	//system("pause");
	//for(j=0;j<numrandomkey;j=j+2)
	breakflag=0;
	for(j=0;j<keynum+1;j++)
	{
		//Flength=0;
			t1=clock();
		for(k=0;k<128;k++)
		{
			for(a=0;a<rownum;a++)
				for(b=0;b<columnum;b++)
				{
						constantterm[a][b] = 0;
						value0[a][b] = 0;
						value1[a][b] = 0;
						value2[a][b] = 0;
						value3[a][b] = 0;
						value4[a][b] = 0;
				}
			for(i=0;i<10;i++)
				key[i]= 0;
			construc_truth_table_dynamic_CutM_32(cube, dim, roundnum,constantterm,rownum,columnum,key,k);//��ֵf(0)
		
			if(j>0)
			{
				for(i=0;i<10;i++)
						key[i] = randomkey[j][i];
				construc_truth_table_dynamic_CutM_32(cube, dim, roundnum,value0,rownum,columnum,key,k);//key1
			}
			for(a=0;a<rownum;a++)
			{
				//��j=0ʱ, ����Ҫ���
				if(j>0)
				{
					for(b=0;b<columnum;b++)
					{
						constantterm[a][b]= constantterm[a][b]^value0[a][b];//��ֵ1
					}
				}
				temp_M=(U64C(0x01)<<(dim-partdim));
				Moebius(constantterm[a], temp_M);
			}
			a=0;
			for(b=0;b<Flength;b++)
			{
				Filter1[k][b>>5]|=((constantterm[a][Fpos[b]>>5]>>(Fpos[b]&0x01f))&0x01)<<(b&0x1f);
			}
		}
		t2=clock();
			printf("%d  %dms\n",j,t2-t1);
		//����, ������洢�ĸ���������ֵ������ֶ�Moebius�任
		k=(0x01<<partdim);
		if(Flength%32==0)
			tt=Flength/32;
		else
			tt=Flength/32+1;
		//ֻ����һ���ֿ��ܻ�������
		for(i=0;i<partdim;i++)
		{
			Sz=0x01L<<i;
			Pos=0;
			while(Pos<k)
			{
				for(b=0;b<Sz;b++)
				{
					//for(a=0;a<Flength/32+1;a++)
					for(a=0;a<tt;a++)
					{
						Filter1[Pos+Sz+b][a]=Filter1[Pos+Sz+b][a]^Filter1[Pos+b][a];
					}
				}
				Pos=Pos+2*Sz;
			}
		}
		//����ÿһ����ѡ��������Զ��, ����x_i��ϵ��
		for(t=0;t<cannum;t++)
		{
			i=canindex[t][0];
			b=canindex[t][1];
			coeffs[t][j]=(Filter1[i][b>>5] >>(b&0x0000001F))&0x01;
		}
		k=0x01<<partdim;
		for(a=0;a<k;a++)
		{
			for(b=0;b<(Flength/32)+1;b++)
			{
				Filter1[a][b]=0;
			}
		}
	}
	Sleep(01);
	fprintf(fp2,"*************************\n");
	//dim+=partdim;
	n=(0x01<<partdim);
	a=0;

	//system("pause");
	n=(0x01<<partdim);
	a=0;
	//��������ʽд���ļ���
	FILE *fpout;
	fpout=fopen(filename,"a+");
	fprintf(fpout, "Mother cube :");

	for(i=0;i<dim;i++)
	{
		fprintf(fpout, "%d,", cube[i]);
	}
	fprintf(fpout,"\n");

	for(t=0;t<cannum;t++)
	{
		printf("Testing %d-th cube",t);
		i=canindex[t][0];
		l=canindex[t][1];
		{
			//b=Fpos[l];
			b=l;
			//flag1=1;
			subdim=0;
			b= (i<<(dim-partdim))|(Fpos[b]);//��λû������
			
			if(1)
			{
				for(j=0;j<dim;j++)
				{
					if(((b>>j)&0x01) ==1)
					{
						fprintf(fp2,"%d,",cube[j]);
						printf("%d,",cube[j]);
						subdim++;
					}
				}
				printf("\n");
				fprintf(fp2,"\n");
				fprintf(Sdim,"%d,",subdim);
				printf("%d ,%d,%d\n",1,i,subdim);
				fprintf(fpout,"subcube: ");
				for(j=0;j<dim;j++)
				{
					if(((b>>j)&0x01) ==1)
					{
						fprintf(fpout,"%d,",cube[j]);
					}
				}
				fprintf(fpout,"\nsubdim=%d     ", subdim);
				fprintf(fpout,"superpoly: ", subdim);
				if(coeffs[t][0]==1)
					fprintf(fpout,"1");
				else
					fprintf(fpout,"0");
				for(j=1;j<81;j++)
				{
					if(coeffs[t][j]==1)
						fprintf(fpout,"+x%d", j-1);
				}
				fprintf(fpout,"\n\n");
			}
		}
	}

	//Sleep(7200000);
	//////38-42
	//for(i=0;i<0;i++)
	
	for(i=0;i<rownum;i++)
	{
		free(constantterm[i]);
		free(value0[i]);
		free(value1[i]);
		free(value2[i]);
		free(value3[i]);
		free(value4[i]);
	}
	free(constantterm);
	free(value0);
	free(value1);
	free(value2);
	free(value3);
	free(value4);
	for(i=0;i<128;i++)
	{
		free(Filter[i]);
		free(Filter1[i]);
		free(sFilter[i]);
		free(sFilter1[i]);
		free(Fcheck[i]);
		free(Fcheck1[i]);
		free(sFcheck[i]);
		free(sFcheck1[i]);
		free(T2Filter[i]);
		free(T2Filter1[i]);
		free(T2Fcheck[i]);
		free(T2Fcheck1[i]);
		//free(Fpos[i]);
	}
	free(Filter);
	free(Filter1);
	free(sFilter);
	free(sFilter1);
	free(Fcheck);
	free(Fcheck1);
	free(sFcheck);
	free(sFcheck1);
	free(T2Fcheck);
	free(T2Fcheck1);
	free(T2Filter);
	free(T2Filter1);
	free(Fpos);
	fclose(fp2);
	fclose(Sdim);
	fclose(fpout);
	return cubenum;
}
void choose_cube(u8 dim, u32 cube[], u8 startindex)
{
   u8 i=0;
   u8 j=0;
   u8 random_number_tab[256] = {41,20,63,67,10,29,66,32,56,27,56,17,10,44,13,32,50,42,37,67,52,35,71,25,40,59,\
25,48,35,79,17,71,63,31,0,79,7,79,38,9,78,19,54,58,0,71,43,37,11,34,32,67,27,\
49,67,19,25,27,62,33,14,20,61,76,7,43,52,20,26,11,26,78,23,79,7,11,24,63,29,12,\
5,50,37,54,66,17,56,65,22,75,15,67,9,43,6,60,27,74,14,10,65,77,58,23,58,38,47,0,\
77,42,30,69,18,43,35,71,69,3,6,4,9,14,5,7,55,26,64,18,27,9,38,4,0,0,57,69,2,65,\
34,21,49,7,2,35,66,25,35,51,79,61,66,41,29,72,52,79,4,62,22,8,15,41,7,62,68,22,38,\
75,20,37,32,5,64,31,71,20,36,63,35,56,34,31,40,71,57,43,15,12,17,70,16,50,26,71,48,\
31,65,77,9,68,79,25,15,0,68,52,7,72,55,77,73,18,46,76,1,41,9,40,49,20,8,77,49,6,72,\
79,30,28,17,54,76,39,67,61,36,32,49,43,79,31,46,70,34,37,33,77,45,5,33,15,11,17,\
74,47,51,45};
      //srand((unsigned int)(time));
   srand((unsigned int)(time(NULL)));
   if(startindex==0)
   {
		cube[i]=random_number_tab[rand()&0xFF];
		startindex = 1;
   }

 for(i=startindex;i<dim;i++)
   {
	cube[i]=random_number_tab[rand()&0xFF];
LOOP: for (j=0;j<i;j++)
	  {
		if (cube[i]==cube[j]) 
		{
         cube[i]=random_number_tab[rand()&0xFF];
		 goto LOOP;
		}
	  }
   }
}




void search_cube_parallel(u32 cube[], u8 dim, u32 roundnum, u8 startindex, u32 totalnum, char filename[])
{
	u32 k,j=0;
	FILE *fp;
	for(k=0;k<totalnum;k++){ printf("%d,",k);
		choose_cube(dim, cube, startindex);
			 //��ӡcube
			 printf("\n");
			 for(j=0;j<dim;j++)
				 printf("%d,",cube[j]);
		firstoutput  = 1;
		fp=fopen(filename,"a+");
		fprintf(fp,"chosen cube %d: ",k);
		for(j=0;j<dim;j++)
		{
			fprintf(fp,"%d,",cube[j]);
		}
		fprintf(fp,"\n");
		fclose(fp);
		//linearity_test_dynamicV6(cube, dim, roundnum,filename);
	}
}


u32 linearity_test_dynamicVV(u32 cube[], u32 dim, u32 roundnum,u32 randomkey[][10],u32 sfkey[][10], u32 sfkey2[][10], u32 randomkeypro[][10], u32 sfkeypro[][10], u32 sfkeypro2[][10],char filename[])
{
	unsigned __int64  i,j,a,b,weight,l,m,n,Sz,Pos,tt=0,temp_M=0,k=0;
	u32 key[10],partdim=7;
	u32 **constantterm, **value0, **value1, **value2,**value3,**value4;//������2427000
	//u32 **Filter1,**Filter, Flength=2827000, **Fcheck,**Fcheck1, **sFilter1,**sFilter,**sFcheck,**sFcheck1;//3492176
	u32 **Filter1,**Filter, Flength=2427000, **Fcheck,**Fcheck1, **sFilter1,**sFilter,**sFcheck,**sFcheck1,**T2Filter1,**T2Filter,**T2Fcheck,**T2Fcheck1;//3492176
	unsigned __int64 	*Fpos;//���ڴ洢λ��
	u32 numrandomkey = 64;  // ����һ��
	//u32 randomkey[32*2][10],sfkey[16*2][10]={0},sfkey2[32][10]={0},tempkey[32][10]={0};//��һ����Կ
	//u32 randomkeypro[64][10]={0},sfkeypro[16*2][10]={0},sfkeypro2[32][10]={0};//�ڶ�����Կ
	u8 breakflag=0;
	clock_t t1,t2;
	//u32 **ANF;//������Ҫ�ų�������
	FILE *fp2,*Sdim;
	partdim=7;
	u32 flag,flag1,subdim,flag2,flag3,flag4,flag5;
	u32 rownum = 1,cubenum=0;
	unsigned __int64  total, total1,total2,total3;
	u32 columnum = U64C(0x01)<<(dim-5-partdim);
	fp2=fopen("CRec.txt","a+");
	Sdim=fopen("subdim.txt","a+");
	n=Flength;
	constantterm = (u32 **)malloc(rownum*sizeof(u32*));
	Filter1=(u32 **) malloc(sizeof(u32*)*128);
	Filter=(u32 **) malloc(sizeof(u32*)*128);//2ά����,���ڱ���ÿһ�����ֺ�����ANF(ɸѡ��)
	Fcheck=(u32 **) malloc(sizeof(u32*)*128);
	Fcheck1=(u32 **) malloc(sizeof(u32*)*128);
	//Fpos=(unsigned __int64 **) malloc(sizeof(unsigned __int64  *)*Flength);
	Fpos=(unsigned __int64 *) malloc(sizeof(unsigned __int64 )*(Flength*32));//���ڱ���ɸѡ�����ĵ������, ����Ҫ����32

	sFilter1=(u32 **) malloc(sizeof(u32*)*128);//����128���ɣ���
	sFilter=(u32 **) malloc(sizeof(u32*)*128);//2ά����,���ڱ���ÿһ�����ֺ�����ANF(ɸѡ��)
	sFcheck=(u32 **) malloc(sizeof(u32*)*128);
	sFcheck1=(u32 **) malloc(sizeof(u32*)*128);

	T2Filter1=(u32 **) malloc(sizeof(u32*)*128);
	T2Filter=(u32 **) malloc(sizeof(u32*)*128);//2ά����,���ڱ���ÿһ�����ֺ�����ANF(ɸѡ��)
	T2Fcheck=(u32 **) malloc(sizeof(u32*)*128);
	T2Fcheck1=(u32 **) malloc(sizeof(u32*)*128);
	for(i=0;i<rownum;i++)
		constantterm[i]=(u32*)malloc(columnum*sizeof(u32));

	for(i=0;i<128;i++)
	{
		Filter1[i]=(u32*)malloc(Flength*sizeof(u32));
		Filter[i]=(u32*)malloc(Flength*sizeof(u32));
		//Fpos[i]=(unsigned __int64 *)malloc(Flength*sizeof(unsigned __int64));//���ȣ�Fposֻ��Ҫ�洢һ�Σ�
		Fcheck[i]=(u32*)malloc(Flength*sizeof(u32));
		Fcheck1[i]=(u32*)malloc(Flength*sizeof(u32));

		sFilter1[i]=(u32*)malloc(Flength*sizeof(u32));
		sFilter[i]=(u32*)malloc(Flength*sizeof(u32));
		sFcheck[i]=(u32*)malloc(Flength*sizeof(u32));
		sFcheck1[i]=(u32*)malloc(Flength*sizeof(u32));

		T2Filter1[i]=(u32*)malloc(Flength*sizeof(u32));
		T2Filter[i]=(u32*)malloc(Flength*sizeof(u32));
		T2Fcheck[i]=(u32*)malloc(Flength*sizeof(u32));
		T2Fcheck1[i]=(u32*)malloc(Flength*sizeof(u32));
	}
	
	value0 = (u32 **)malloc(rownum*sizeof(u32*));
	value1 = (u32 **)malloc(rownum*sizeof(u32*));
	value2 = (u32 **)malloc(rownum*sizeof(u32*));
	value3 = (u32 **)malloc(rownum*sizeof(u32*));
	value4 = (u32 **)malloc(rownum*sizeof(u32*));
	for(i=0;i<rownum;i++)
	{
		value0[i]=(u32*)malloc(columnum*sizeof(u32));
		value1[i]=(u32*)malloc(columnum*sizeof(u32));
		value2[i]=(u32*)malloc(columnum*sizeof(u32));
		value3[i]=(u32*)malloc(columnum*sizeof(u32));
		value4[i]=(u32*)malloc(columnum*sizeof(u32));
	}


	//�������һ��ѭ��
	partdim=7;
	//partdim=7;
	
	//����һ�������Կ
	//srand((unsigned int) time(NULL));
	//for(i=0;i<numrandomkey;i++)
	//	choose_random_key(randomkey[i]);//GenRandomKeyV3Right(randomkey,sfkey2);
	////genrandomkeyV2(randomkey,sfkey);//��������һ�������Կ, ̫��,00.06
	//genrandomkeyV2Right(randomkey,sfkey);
	//printf("\n%d\n",randomkey[63][0]);
	//printf("%d\n",sfkey[31][0]);
	//printf("RandomKey1  DONE >>>>\n");
	//GenRandomKeyV3Right(randomkey,sfkey2);
	//printf("%d\n",sfkey2[31][0]);
	//printf("RandomKey2  DONE >>>>\n");
	////dim-=partdim;
	//genrandomkeyV2Right(randomkeypro,sfkeypro);
	//GenRandomKeyV3Right(randomkeypro,sfkeypro2);
	for(a=0;a<128;a++)
		for(b=0;b<Flength;b++)
		{
			Filter[a][b]=0;
			Filter1[a][b]=0;
			Fpos[b]=0;
			Fcheck[a][b]=0;
			Fcheck1[a][b]=0;

			sFilter[a][b]=0;
			sFilter1[a][b]=0;
			sFcheck[a][b]=0;
			sFcheck1[a][b]=0;

			T2Filter[a][b]=0;
			T2Filter1[a][b]=0;
			T2Fcheck[a][b]=0;
			T2Fcheck1[a][b]=0;
		}
	//������Ҫ�����õ����������������ʱ����Ҫ
	Flength=0;
	m=U64C(0x01)<<(dim-partdim);
	printf("Start get index>>>>\n");
	for(b=0;b<m;b++)
	{
		weight=0;
		l=b;
		for(i=0;i<dim;i++)
		{
			weight+=(l&0x01);
			l>>=1;
		}
		if(weight>(dim-partdim-9))//41-7-10+1=25+7=32//cong32kaishi
		//if(weight>21)//42-9+1//ֱ��д�����֣�Ȼ������֤һ���Ƿ���ȷ
		{
			Fpos[Flength]= (U64C(0x00)<<(dim-partdim))|b;
			Flength++;
		}
	}
		//�ܹ���ҪFlength���㣬ÿ������һ���Ӻ�������ֵ���Ժ�ɸȡ��Ӧλ�õ�0��1���뵽Filter�Ȼ���Filter����Moebius�任
	printf("Done>>>>\nStart  Computing Cube>>>>\n");
	printf("%u\n",Flength);
	printf("%llu\n",Fpos[Flength-1]);
	printf("%llu\n",Fpos[Flength-2]);
	printf("%llu\n",Fpos[Flength-3]);
	//system("pause");
	//for(j=0;j<numrandomkey;j=j+2)
	breakflag=0;
	for(j=0;j<10;j=j+2)
	{
		//Flength=0;
			t1=clock();
		for(k=0;k<128;k++)
		{
		//	Flength=0;
		//
			for(a=0;a<rownum;a++)
				for(b=0;b<columnum;b++)
				{
						constantterm[a][b] = 0;
						value0[a][b] = 0;
						value1[a][b] = 0;
						value2[a][b] = 0;
						value3[a][b] = 0;
						value4[a][b] = 0;
				}
			//ȡcube,Ȼ����㳣ֵ
		for(i=0;i<10;i++)
			key[i]= 0;
		construc_truth_table_dynamic_CutM_32(cube, dim, roundnum,constantterm,rownum,columnum,key,k);//��ֵf(0)
			
		for(i=0;i<10;i++)
				key[i] = randomkey[j][i];
		construc_truth_table_dynamic_CutM_32(cube, dim, roundnum,value0,rownum,columnum,key,k);//key1

		for(i=0;i<10;i++)
				key[i]=randomkey[j+1][i];
		construc_truth_table_dynamic_CutM_32(cube, dim, roundnum,value3,rownum,columnum,key,k);//key2��constantterm1-��value1�� value0ʵ����f(key1)+f(key2)
			
		for(i=0;i<10;i++)
				key[i]=randomkey[j][i]^randomkey[j+1][i];
		construc_truth_table_dynamic_CutM_32(cube, dim, roundnum,value1,rownum,columnum,key,k);//key1^key2 f(key1+key2)


		//����
		//for(i=0;i<10;i++)
				//key[i]=sfkey[j/2][i];//����������
		//construc_truth_table_dynamic_CutM_32(cube, dim, roundnum,value2,rownum,columnum,key,k);//f(key1'+key2')


		for(i=0;i<10;i++)
				key[i]=sfkey2[j/2][i];//����������
		construc_truth_table_dynamic_CutM_32(cube, dim, roundnum,value4,rownum,columnum,key,k);//f(key1''+key2'')

			for(a=0;a<rownum;a++){
				for(b=0;b<columnum;b++)
				{
					value1[a][b] = constantterm[a][b]^value1[a][b]^value0[a][b]^value3[a][b];//������ʽ1
					//value2[a][b] = constantterm[a][b]^value2[a][b]^value0[a][b]^value3[a][b];//���Լ��-��ʽ2
					//value4[a][b] = constantterm[a][b]^value4[a][b]^value0[a][b]^value3[a][b];//newtype2
					value0[a][b]= constantterm[a][b]^value0[a][b];//��ֵ1
					//constantterm[a][b]= constantterm[a][b]^value0[a][b];//��ֵ2
				}
				temp_M=(U64C(0x01)<<(dim-partdim));
				Moebius(value1[a], temp_M);
				//Moebius(value2[a], temp_M);
				//Moebius(value0[a], temp_M);
				Moebius(value4[a], temp_M);
				Moebius(constantterm[a], temp_M);
			 }
			a=0;
			for(b=0;b<Flength;b++)
			{
				Filter[k][b>>5]|=((value1[a][Fpos[b]>>5]>>(Fpos[b]&0x01f))&0x01)<<(b&0x1f);//ÿһ������ʽ��ϵ��ռ��һ�����أ�Ȼ��������е�
				Filter1[k][b>>5]|=((value0[a][Fpos[b]>>5]>>(Fpos[b]&0x01f))&0x01)<<(b&0x1f);
				//
				sFilter[k][b>>5]|=((value2[a][Fpos[b]>>5]>>(Fpos[b]&0x01f))&0x01)<<(b&0x1f);
				sFilter1[k][b>>5]|=((constantterm[a][Fpos[b]>>5]>>(Fpos[b]&0x01f))&0x01)<<(b&0x1f);
				//newtype
				T2Filter[k][b>>5]|=((value4[a][Fpos[b]>>5]>>(Fpos[b]&0x01f))&0x01)<<(b&0x1f);
				T2Filter1[k][b>>5]|=((value0[a][Fpos[b]>>5]>>(Fpos[b]&0x01f))&0x01)<<(b&0x1f);
				//Fpos[Flength]=(b<<4)|k;
				//(k<<(dim-partdim))|b;
				//	Flength++;
			}
		}
		t2=clock();
			printf("%d  %dms\n",j,t2-t1);
		//����, ������洢�ĸ���������ֵ������ֶ�Moebius�任
		k=(0x01<<partdim);
		if(Flength%32==0)
			tt=Flength/32;
		else
			tt=Flength/32+1;
		//ֻ����һ���ֿ��ܻ�������
		for(i=0;i<partdim;i++)
		{
			Sz=0x01L<<i;
			Pos=0;
			while(Pos<k)
			{
				for(b=0;b<Sz;b++)
				{
					//for(a=0;a<Flength/32+1;a++)
					for(a=0;a<tt;a++)
					{
						Filter[Pos+Sz+b][a]=Filter[Pos+Sz+b][a]^Filter[Pos+b][a];//ֻҪλ�����໥��Ӧ�ľͿ��ԣ��Ȼ���һ����һ��
						Filter1[Pos+Sz+b][a]=Filter1[Pos+Sz+b][a]^Filter1[Pos+b][a];
						//
						//
						sFilter[Pos+Sz+b][a]=sFilter[Pos+Sz+b][a]^sFilter[Pos+b][a];
						sFilter1[Pos+Sz+b][a]=sFilter1[Pos+Sz+b][a]^sFilter1[Pos+b][a];
						//
						//
						T2Filter[Pos+Sz+b][a]=T2Filter[Pos+Sz+b][a]^T2Filter[Pos+b][a];
						T2Filter1[Pos+Sz+b][a]=T2Filter1[Pos+Sz+b][a]^T2Filter1[Pos+b][a];
					}
				}
				Pos=Pos+2*Sz;
			}
		}
		if(Flength%32==0)
			tt=Flength/32;
		else
			tt=Flength/32+1;
		for(a=0;a< U64C(0x01)<<(partdim);a++)
		{
			for(k=0;k<tt;k++)
			{
				Fcheck[a][k]|=Filter[a][k];
				Fcheck1[a][k]|=Filter1[a][k];

				sFcheck[a][k]|=sFilter[a][k];
				sFcheck1[a][k]|=sFilter1[a][k];

				T2Fcheck[a][k]|=T2Filter[a][k];
				T2Fcheck1[a][k]|=T2Filter1[a][k];
			}
		}
		k=0x01<<partdim;
		for(a=0;a<k;a++)
		{
			for(b=0;b<(Flength/32)+1;b++)
			{
				Filter[a][b]=0;
				Filter1[a][b]=0;
				sFilter[a][b]=0;
				sFilter1[a][b]=0;
				T2Filter[a][b]=0;
				T2Filter1[a][b]=0;
			}
		}
		if((j%16==0)&&(j>0))
			Sleep(01);
	}
	Sleep(01);
	fprintf(fp2,"*************************\n");
	//dim+=partdim;
	n=(0x01<<partdim);
	a=0;
	for(i=0;i<n;i++)//��ȷ
	{
		printf("%d,",i);
		for(l=0;l<Flength;l++)
		{
			//b=Fpos[l];
			b=l;
			flag = Fcheck[i][b>>5] & ((U64C(0x01))<<(b&0x0000001F));
			flag1 = Fcheck1[i][b>>5] & ((U64C(0x01))<<(b&0x0000001F));

			flag2= sFcheck[i][b>>5] & ((U64C(0x01))<<(b&0x0000001F));// (sFcheck[i][b>>5]>>(b&0x0000001F))&0x01;// & ((U64C(0x01))<<(b&0x0000001F));
			flag3= Fcheck1[i][b>>5] & ((U64C(0x01))<<(b&0x0000001F));//(sFcheck1[i][b>>5]>>(b&0x0000001F))&0x01;

			flag4= T2Fcheck[i][b>>5] & ((U64C(0x01))<<(b&0x0000001F));// (sFcheck[i][b>>5]>>(b&0x0000001F))&0x01;// & ((U64C(0x01))<<(b&0x0000001F));
			flag5= Fcheck1[i][b>>5] & ((U64C(0x01))<<(b&0x0000001F));//(sFcheck1[i][b>>5]>>(b&0x0000001F))&0x01;
			//flag1=1;
			subdim=0;
			b= (i<<(dim-partdim))|(Fpos[b]);//��λû������
			if(((flag==0)&&(flag1!=0)))
			{
				fprintf(fp2,"****Type1*****");
				printf("****Type1*****");
				for(j=0;j<dim;j++)
				{
					if(((b>>j)&0x01) ==1)
					{
						fprintf(fp2,"%d,",cube[j]);
						printf("%d,",cube[j]);
						subdim++;
					}
				}
				printf("\n");
				fprintf(fp2,"\n");
				fprintf(Sdim,"%d,",subdim);
				//printf("%d ,%d,%d\n",1,i,subdim);
				//outputsubcubes(cube,dim,roundnum,a,b,filename);
			}
			///
			subdim=0;
			//if(((flag2==0)&&(flag3!=0)))
			if(0)
			{
				fprintf(fp2,"****Type2*****");
				printf("****Type2*****");
				for(j=0;j<dim;j++)
				{
					if(((b>>j)&0x01) ==1)
					{
						fprintf(fp2,"%d,",cube[j]);
						printf("%d,",cube[j]);
						subdim++;
					}
				}
				printf("\n");
				fprintf(fp2,"\n");
				//printf("%d,%d,%d\n",2,i,subdim);

				//genrandomkeyV2Right(randomkey,sfkey);
				//outputcubeofsf(cube,dim,roundnum,a,b,filename,randomkeypro,sfkeypro);
			}

			subdim=0;
			//if(((flag4==0)&&(flag5!=0)))
			if(0)
			{
				fprintf(fp2,"****Type3*****");
				printf("****Type3*****");
				for(j=0;j<dim;j++)
				{
					if(((b>>j)&0x01) ==1)
					{
						fprintf(fp2,"%d,",cube[j]);
						printf("%d,",cube[j]);
						subdim++;
					}
				}
				printf("\n");
				fprintf(fp2,"\n");
				//printf("%d,%d,%d\n",3,i,subdim);

				//srand((unsigned int) time(NULL));
				/*for(i=0;i<numrandomkey;i++)
					choose_random_key(randomkey[i]);
				GenRandomKeyV3Right(randomkey,sfkey2);*/
				//outputcubeofsf2(cube,dim,roundnum,a,b,filename,randomkeypro,sfkeypro2);
			}
		}
	}
	printf("\nDone1>>>>\n");
	//system("pause");
	n=(0x01<<partdim);
	a=0;
	for(i=0;i<n;i++)//��ȷ
	{
		printf("%d,",i);
		for(l=0;l<Flength;l++)
		{
			//b=Fpos[l];
			b=l;
			flag = Fcheck[i][b>>5] & ((U64C(0x01))<<(b&0x0000001F));
			flag1 = Fcheck1[i][b>>5] & ((U64C(0x01))<<(b&0x0000001F));

			flag2= sFcheck[i][b>>5] & ((U64C(0x01))<<(b&0x0000001F));// (sFcheck[i][b>>5]>>(b&0x0000001F))&0x01;// & ((U64C(0x01))<<(b&0x0000001F));
			flag3= Fcheck1[i][b>>5] & ((U64C(0x01))<<(b&0x0000001F));//(sFcheck1[i][b>>5]>>(b&0x0000001F))&0x01;

			flag4= T2Fcheck[i][b>>5] & ((U64C(0x01))<<(b&0x0000001F));// (sFcheck[i][b>>5]>>(b&0x0000001F))&0x01;// & ((U64C(0x01))<<(b&0x0000001F));
			flag5= Fcheck1[i][b>>5] & ((U64C(0x01))<<(b&0x0000001F));//(sFcheck1[i][b>>5]>>(b&0x0000001F))&0x01;
			//flag1=1;
			subdim=0;
			b= (i<<(dim-partdim))|(Fpos[b]);//��λû������
			if(((flag==0)&&(flag1!=0)))
			{
				for(j=0;j<dim;j++)
				{
					if(((b>>j)&0x01) ==1)
					{
						fprintf(fp2,"%d,",cube[j]);
						printf("%d,",cube[j]);
						subdim++;
					}
				}
				printf("\n");
				fprintf(fp2,"\n");
				fprintf(Sdim,"%d,",subdim);
				printf("%d ,%d,%d\n",1,i,subdim);
				outputsubcubes(cube,dim,roundnum,a,b,filename);
			}
			///
			subdim=0;
			if(((flag2==0)&&(flag3!=0)))
			{
				fprintf(fp2,"****Type2*****");
				printf("****Type2*****");
				for(j=0;j<dim;j++)
				{
					if(((b>>j)&0x01) ==1)
					{
						fprintf(fp2,"%d,",cube[j]);
						printf("%d,",cube[j]);
						subdim++;
					}
				}
				printf("\n");
				fprintf(fp2,"\n");
				printf("%d,%d,%d\n",2,i,subdim);

				//genrandomkeyV2Right(randomkey,sfkey);
				outputcubeofsf(cube,dim,roundnum,a,b,filename,randomkeypro,sfkeypro);
			}

			subdim=0;
			if(((flag4==0)&&(flag5!=0)))
			{
				fprintf(fp2,"****Type3*****");
				printf("****Type3*****");
				for(j=0;j<dim;j++)
				{
					if(((b>>j)&0x01) ==1)
					{
						fprintf(fp2,"%d,",cube[j]);
						printf("%d,",cube[j]);
						subdim++;
					}
				}
				printf("\n");
				fprintf(fp2,"\n");
				printf("%d,%d,%d\n",3,i,subdim);

				srand((unsigned int) time(NULL));
				/*for(i=0;i<numrandomkey;i++)
					choose_random_key(randomkey[i]);
				GenRandomKeyV3Right(randomkey,sfkey2);*/
				outputcubeofsf2(cube,dim,roundnum,a,b,filename,randomkeypro,sfkeypro2);
			}
		}
	}
	//Sleep(7200000);
	//////38-42
	//for(i=0;i<0;i++)

	for(i=0;i<rownum;i++)
	{
		free(constantterm[i]);
		free(value0[i]);
		free(value1[i]);
		free(value2[i]);
		free(value3[i]);
		free(value4[i]);
	}
	free(constantterm);
	free(value0);
	free(value1);
	free(value2);
	free(value3);
	free(value4);
	for(i=0;i<128;i++)
	{
		free(Filter[i]);
		free(Filter1[i]);
		free(sFilter[i]);
		free(sFilter1[i]);
		free(Fcheck[i]);
		free(Fcheck1[i]);
		free(sFcheck[i]);
		free(sFcheck1[i]);
		free(T2Filter[i]);
		free(T2Filter1[i]);
		free(T2Fcheck[i]);
		free(T2Fcheck1[i]);
		//free(Fpos[i]);
	}
	free(Filter);
	free(Filter1);
	free(sFilter);
	free(sFilter1);
	free(Fcheck);
	free(Fcheck1);
	free(sFcheck);
	free(sFcheck1);
	free(T2Fcheck);
	free(T2Fcheck1);
	free(T2Filter);
	free(T2Filter1);
	free(Fpos);
	fclose(fp2);
	fclose(Sdim);
	return cubenum;
}

 ///ֻ��Ҫ����������м���,�ֲ�����,�������豸��ʵ��
 //��Ҫ��ÿһ���߳����ظ�װ����Կ


//��������Ҫ�����Լ��ĳ���host�˳��������е���device�˵���ͳ���

void getIndex(int **canindex)
{
	FILE *fp;
	fopen_s(&fp, "candiind", "r");
	//
	int i,j,m=0,l=0;
	int cubenum = 838;
	char c;
	char buff[1000] = { 0 };
	c = fgetc(fp);
	char X[80] = { 0 };
	int recround[10] = { 0 };
	int dim = 0;
	int maxround = 0;
	while (c != EOF)
	{
		//
		memset(buff, 0, sizeof(char) * 1000);
		j = 0;
		while (c != '\n')
		{
			buff[j++] = c;
			c = fgetc(fp);
		}
		c = fgetc(fp);
		m = 0;
		i = 0;
		int flag=0;
		//printf("%d: ", l);
		while(i<j)
		//for (i = 0; i < j; i++)
		{
			if ((buff[i]>='0') && (buff[i]<='9'))
			{
				flag=0;
				if (buff[i + 1] == ',' && flag==0)
				{
					canindex[l][m++] = buff[i] - '0';
					i = i + 2;
					flag=1;
				}
				if (buff[i + 2] == ',' &&flag==0)
				{
					canindex[l][m++] = (buff[i] - '0')*10+(buff[i+1] - '0');
					i = i + 3;
					flag=1;
				}
				if (buff[i + 3] == ',' &&flag==0)
				{
					canindex[l][m++] = (buff[i] - '0')*100+(buff[i+1] - '0')*10+(buff[i+2] - '0');
					i = i + 4;
					flag=1;
				}
				else
				{
					if(flag==0)
					{
						canindex[l][m]=0;
						int tmp=j-i-1,tt;
						int ti=1;
						//while(i<j)
						for(tt=0;tt<tmp;tt++)
						{
							canindex[l][m]+=(buff[j-2-tt]-'0')*ti;
							ti*=10;
						}
						m++;
						i=j;
					}
				}
			}
		}
		//cancubedim[l]=m;
		l++;
	}
	fclose(fp);
}
//�̲߳���

int main()
{
	clock_t t1,t2;
	u32 roundnum;
	u32 dim;
	roundnum=805;
	int i;
	dim=40;
	//u32 cube[40]={4,6,10,11,25,17,19,21,25,29,32,34,36,39,41,43,50,2,70,0,15,8,27,26,79,1,13,28,45,38,23,9,47,76,67,24,42,57,71,72};
	u32 cube[40]={ 2,4,6,10,11,12,15,17,19,21,23,25,29,34,36,41,0,70,8,16,79,27,45,26,28,31,77,38,47,13,1,62,49,64,40,39,43,50,58,74 };
	//u32 cube[31]={0,2,4,6,8,10,13,15,17,19,21,23,25,27,28,29,32,34,36,38,39,41,43,45,47,48,53,69,71,75,79,};//dim 31 for 805-round
	//u32 cube[22]={77,73,71,70,60,56,55,50,47,46,45,37,32,30,28,27,26,22,18,15,6,2};
	int dimlist[12]={21,21,21,22,22,22,23,23,23,24};
	u32 totalnum;
	u8 lin_test[32];
	//printf("Enter roundnum and totalnum\n");
	//scanf("%d, %d, %d", &roundnum, &totalnum,&dim);
	//printf("%u,%u,%u\n",roundnum,dim,totalnum);
	char filename[40];
	sprintf(filename,"result_round(%d)_dim(%d)",roundnum,dim);
	int **canindex;
	int candinum=838;
	canindex=(int**)malloc(sizeof(int*)*candinum);
	for(i=0;i<candinum;i++)
	{
		canindex[i]=(int *)malloc(sizeof(int)*2);
		memset(canindex[i],0,sizeof(int)*2);
	}
	getIndex(canindex);
	for(i=0;i<1;i++)
	{
		printf("%d %u\n", canindex[i][0],canindex[i][1]);
	}
	t1=clock();
	//GetSum();
	for(i=0;i<1;i++)
	{
		printf("\n***********%d*************\n",i);
		linearity_test_dynamicV6(cube,dim,roundnum,canindex,candinum,filename);
	}
	t2=clock();
	for(i=0;i<candinum;i++)
		free(canindex[i]);
	free(canindex);
	printf("\n%dms\n",t2-t1);
	system("pause");
	return 0;
}